#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "Network_cuda.h"

//  lldb print commands -- use memory read with format
//  me r -rf f -l148 -c148 send_netin_tmp_h

Network_cuda::Network_cuda() {
  Initialize();
}

Network_cuda::~Network_cuda() {
  FreeCudaArrays();
}

void Network_cuda::Initialize() {
  strms_created = false;

  con_group_size = 0;
  unit_vars_size = 0;
  n_units_built = 0;
  n_layers_built = 0;
  n_ungps_built = 0;
  units_mem_h = NULL;
  units_mem_d = NULL;
  n_lay_stats = 0;
  n_lay_stats_vars = 0;
  lay_stats_h = NULL;
  lay_stats_d = NULL;
  units_n_recv_cgps_h = NULL;
  units_n_recv_cgps_d = NULL;
  units_n_send_cgps_h = NULL;
  units_n_send_cgps_d = NULL;
  n_recv_cgps = 0;
  n_send_cgps = 0;
  recv_cgp_mem_h = NULL;
  recv_cgp_mem_d = NULL;
  send_cgp_mem_h = NULL;
  send_cgp_mem_d = NULL;
  recv_cgp_start_h = NULL;
  recv_cgp_start_d = NULL;
  send_cgp_start_h = NULL;
  send_cgp_start_d = NULL;
  recv_cons_cnt = 0;
  send_cons_cnt = 0;
  recv_cons_mem_h = NULL;
  recv_cons_mem_d = NULL;
  send_cons_mem_h = NULL;
  send_cons_mem_d = NULL;
  own_cons_max_size = 0;
  own_cons_tot_size = 0;
  own_cons_tot_size_nonshared = 0;
  own_cons_avg_size = 0;
  own_cons_max_vars;

  con_params_h = NULL;
  con_params_d = NULL;
}

void Network_cuda::FreeCudaArrays() {
  if(recv_cons_mem_d)
    hipFree(recv_cons_mem_d);
  if(send_cons_mem_d)
    hipFree(send_cons_mem_d);

  if(recv_cgp_mem_d)
    hipFree(recv_cgp_mem_d);
  if(send_cgp_mem_d)
    hipFree(send_cgp_mem_d);
  if(recv_cgp_start_mem_d)
    hipFree(recv_cgp_start_mem_d);
  if(send_cgp_start_mem_d)
    hipFree(send_cgp_start_mem_d);
  
  if(units_n_recv_cgps_d)
    hipFree(units_n_recv_cgps_d);
  if(units_n_send_cgps_d)
    hipFree(units_n_send_cgps_d);

  if(lay_stats_d)
    hipFree(lay_stats_d);
  
  if(units_mem_d)
    hipFree(units_mem_d);

  if(con_params_h)
    free(con_params_h);
  if(con_params_d)
    hipFree(con_params_d);

   if(strms_created) {
    hipStreamDestroy(strm_memcpy_cons);
    hipStreamDestroy(strm_memcpy_units);
    hipStreamDestroy(strm_compute_netin);
    hipStreamDestroy(strm_compute_dwt);
    hipStreamDestroy(strm_compute_wt);
    strms_created = false;
  }

  Initialize();
}

void Network_cuda::AllocCudaArrays
( int    cgs,
  int    uvs,
  int    nub,
  int    nlb,
  int    nugb,
  char*  umh,

  int    nls,
  int    nlsv,
  float* lsh,
  int*   unrch,
  int*   unsch,
  
  int    nrcg,
  int    nscg,

  char*  rcgmh,
  char*  scgmh,
  int*   rcsh,
  int*   scsh,

  bigint rcc,
  bigint scc,
  float* rcmh,
  float* scmh
  )
{
  if(!strms_created) {
    hipStreamCreate(&strm_memcpy_cons);
    hipStreamCreate(&strm_memcpy_units);
    hipStreamCreate(&strm_compute_netin);
    hipStreamCreate(&strm_compute_act);
    hipStreamCreate(&strm_compute_dwt);
    hipStreamCreate(&strm_compute_wt);
    strms_created = true;
  }
  
  min_threads = min_th;
  max_threads = max_th;
  cons_per_thread = cons_per_th;

  // docs on number of threads: http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#execution-configuration-optimizations

  n_threads = (int)round((float)own_cons_avg_size / (float)cons_per_thread);
  int mod32 = n_threads % 32;
  if(mod32 > 15)                // round up
    n_threads = ((n_threads / 32) + 1) * 32;
  else
    n_threads = (n_threads / 32) * 32;
  if(n_threads < min_threads)
    n_threads = min_threads;
  if(n_threads > max_threads)
    n_threads = max_threads;

  con_group_size = cgs;
  unit_vars_size = uvs;
  n_units_built = nub;
  n_layers_built = nlb;
  n_ungps_built = nugb;

  units_mem_h = umh;
  cudaSafeCall(hipMalloc(&units_mem_d, n_units_built * sizeof(char)));

  n_lay_stats = nls;
  n_lay_stats_vars = nlsv;
  lay_stats_h = lsh;
  cudaSafeCall(hipMalloc(&lay_stats_d, n_lay_stats * n_lay_stats_vars * n_layers_built * sizeof(float)));

  units_n_recv_cgps_h = unrch;
  cudaSafeCall(hipMalloc(&units_n_recv_cgps_d, n_units_built * sizeof(int)));
  units_n_send_cgps_h = unsch;
  cudaSafeCall(hipMalloc(&units_n_send_cgps_d, n_units_built * sizeof(int)));

  n_recv_cgps = nrcg;
  n_send_cgps = nscg;

  recv_cgp_mem_h = rcgmh;
  cudaSafeCall(hipMalloc(&recv_cgp_mem_d, n_recv_cgps * con_group_size * sizeof(char)));
  send_cgp_mem_h = scgmh;
  cudaSafeCall(hipMalloc(&send_cgp_mem_d, n_send_cgps * con_group_size * sizeof(char)));
  
  recv_cgp_start_h = rcsh;
  cudaSafeCall(hipMalloc(&recv_cgp_start_d, n_units_built * sizeof(int)));
  send_cgp_start_h = scsh;
  cudaSafeCall(hipMalloc(&send_cgp_start_d, n_units_built * sizeof(int)));

  recv_cons_cnt = rcc;
  send_cons_cnt = scc;
  
  recv_cons_mem_h = rcmh;
  cudaSafeCall(hipMalloc(&recv_cons_mem_d, recv_cons_cnt * sizeof(float)));
  send_cons_mem_h = scmh;
  cudaSafeCall(hipMalloc(&send_cons_mem_d, send_cons_cnt * sizeof(float)));
  
  con_params_h = (float*)malloc(n_recv_cgps * N_CON_PARAMS * sizeof(float));
  cudaSafeCall(hipMalloc(&con_params_d, n_recv_cgps * N_CON_PARAMS * sizeof(float)));
}

void Network_cuda::RecvCons_HostToDevice(bool sync) {
  if(recv_cons_mem_h && recv_coms_mem_d) {
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_d, recv_cons_mem_h,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyHostToDevice, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::RecvCons_DeviceToHost(bool sync) {
  if(recv_cons_mem_h && recv_cons_mem_d) {
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_h, recv_cons_mem_d,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyDeviceToHost, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::UpdateUnitsXCons() {
  if(!con_mem_idxs_h) return;

  int sz =  recv_units_x_cons * sizeof(int);

  cudaSafeCall(hipMemcpy(units_d, units_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_mem_idxs_d, con_mem_idxs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_allocs_d, con_allocs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_sizes_d, con_sizes_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_recv_idxs_d, con_recv_idxs_h, sz, hipMemcpyHostToDevice));
}

void Network_cuda::UpdateConParams() {
  if(!con_params_h) return;

  int sz =  own_units_x_cons * N_CON_PARAMS * sizeof(float);

  cudaSafeCall(hipMemcpy(con_params_d, con_params_h, sz, hipMemcpyHostToDevice));

  if(wt_sig_fun_h) {
    cudaSafeCall(hipMemcpy(wt_sig_fun_d, wt_sig_fun_h, 10002 * sizeof(float),
                            hipMemcpyHostToDevice));
  }
}

__global__ void Kernel_Compute_Netin
(int* cur_units_x_cons_d, float* send_net_acts_d, float* compute_netin_tmp_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const float send_eff = send_net_acts_d[csni];
  const int sz = con_sizes_d[ucidx];
  const float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::WT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  const int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    atomicAdd(&(compute_netin_tmp_d[ridx]), wts[st] * send_eff);
    // compute_netin_tmp_d[ridx] += wts[st] * send_eff; // determine effect of atomic -- not much penalty there
    st++;
  }
}

void Network_cuda::Compute_Netin() {
  if(cur_units_x_cons_n == 0) return;

  // cudaSafeCall(hipMemsetAsync(compute_netin_tmp_d, 0,
  //                              (n_units+1) * send_net_max_prjns * sizeof(float),
  //                              strm_compute_netin));

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_compute_netin));
  cudaSafeCall(hipMemcpyAsync(send_net_acts_d, send_net_acts_h,
                               cur_units_x_cons_n * sizeof(float),
                               hipMemcpyHostToDevice, strm_compute_netin));

  //  Invoke kernel
  Kernel_Compute_NetinDelta<<<cur_units_x_cons_n, n_threads, 0, strm_compute_netin>>>
    (cur_units_x_cons_d, send_net_acts_d, compute_netin_tmp_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d);

  cudaSafeCall(hipMemcpyAsync(compute_netin_tmp_h, compute_netin_tmp_d,
                               (n_units+1) * send_net_max_prjns * sizeof(float),
                               hipMemcpyDeviceToHost, strm_compute_netin));
  // get results back from device -- args are reversed here!

  cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
}


__global__ void Kernel_Compute_dWt
(int* cur_units_x_cons_d, float* unit_vec_vars_d, float* con_params_d, int* units_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 const int nu) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const int sidx = units_d[ucidx];

  const float su_avg_s = unit_vec_vars_d[Network_cuda::AVG_S * nu + sidx];
  const float su_avg_m = unit_vec_vars_d[Network_cuda::AVG_M * nu + sidx];

  const float s_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
                                   Network_cuda::S_MIX];
  const float m_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
                                   Network_cuda::M_MIX];
  const float thr_l_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
                                       Network_cuda::THR_L_MIX];
  const float thr_max = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
                                     Network_cuda::THR_MAX];
  const float clrate = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
                                    Network_cuda::CUR_LRATE];

  const int sz = con_sizes_d[ucidx];
  float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::DWT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    const float ru_avg_s = unit_vec_vars_d[Network_cuda::AVG_S * nu + ridx];
    const float ru_avg_m = unit_vec_vars_d[Network_cuda::AVG_M * nu + ridx];
    const float ru_avg_l = unit_vec_vars_d[Network_cuda::AVG_L * nu + ridx];

    // unfortunately, cos_diff_lmix is on recv layer -- so this needs to be in the loop
    // whereas normally it is outside the loop.. would require a separate
    // var array just for it, at the unit_x_con level..  could look into it later
    const float cos_diff_lmix = unit_vec_vars_d[Network_cuda::COS_DIFF_LMIX * nu
                                                + ridx];
    const float efflmix = thr_l_mix * cos_diff_lmix;
    const float effmmix = 1.0f - efflmix;
    const float su_act_mult = efflmix * su_avg_m;

    const float srs = ru_avg_s * su_avg_s;
    const float srm = ru_avg_m * su_avg_m;
    const float sm_mix = s_mix * srs + m_mix * srm;
    const float lthr = su_act_mult * ru_avg_l;
    float effthr = effmmix * srm + lthr;
    effthr = effthr < thr_max ? effthr : thr_max; // max = thr_max

    float rval;                 // xcal.dWtFun
    if(sm_mix < 0.0001f)        // d_thr = 0.0001
      rval = 0.0f;
    else if(sm_mix > effthr * 0.1f) // d_rev = 0.1
      rval = (sm_mix - effthr);
    else
      rval = sm_mix * -9.0f;    // d_rev_ratio = -9.0;
    dwts[st] += clrate * rval;
    st++;
  }
}

void Network_cuda::Compute_dWt(bool sync) {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_compute_dwt));
  cudaSafeCall(hipMemcpyAsync(unit_vec_vars_d, unit_vec_vars_h,
                               (n_units+1) * N_VEC_VARS * sizeof(float),
                               hipMemcpyHostToDevice, strm_compute_dwt));

  //  Invoke kernel
  Kernel_Compute_dWt_cosdif<<<cur_units_x_cons_n, n_threads, 0, strm_compute_dwt>>>
    (cur_units_x_cons_d, unit_vec_vars_d, con_params_d, units_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, n_units+1);

  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
  }
}

__global__ void Kernel_Compute_Weights
(float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 float* wt_sig_fun_d) {
  const int ucidx = blockIdx.x;  // full unit x con idx here
  const int nth = blockDim.x;

  const int sz = con_sizes_d[ucidx];
  float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::WT));
  float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::DWT));
  float* fwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::FWT));
  float* swts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + Network_cuda::SWT));

  int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    float& dwt = dwts[st];
    if(dwt != 0.0f) {
      float& wt = wts[st];
      float& fwt = fwts[st];
      float& swt = swts[st];
      if(dwt > 0.0f)  dwt *= (1.0f - fwt);
      else            dwt *= fwt;
      fwt += dwt;
      swt = fwt;                // keep sync'd -- not tech necc..

      int idx = __float2int_rd(fwt * 10000.0f); // sig_res_inv
      wt = wt_sig_fun_d[idx];

      dwt = 0.0f;
    }
    st++;
  }
}

void Network_cuda::Compute_Weights(bool sync) {
  //  Invoke kernel -- does all
  Kernel_Compute_Weights<<<own_units_x_cons, n_threads, 0, strm_compute_wt>>>
    (own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, wt_sig_fun_d);

  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_compute_wt));
  }
}



