#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "Network_cuda.h"

//  lldb print commands -- use memory read with format
//  me r -rf f -l148 -c148 send_netin_tmp_h

Network_cuda::Network_cuda() {
  Initialize();
}

Network_cuda::~Network_cuda() {
  NetFree();
}

void Network_cuda::Initialize() {
  strms_created = false;

  con_group_size = 0;
  unit_vars_size = 0;
  n_units_built = 0;
  n_layers_built = 0;
  n_ungps_built = 0;
  units_mem_h = NULL;
  units_mem_d = NULL;
  n_lay_stats = 0;
  n_lay_stats_vars = 0;
  lay_stats_h = NULL;
  lay_stats_d = NULL;
  recv_owns_cons = true;
  units_n_recv_cgps_h = NULL;
  units_n_recv_cgps_d = NULL;
  units_n_send_cgps_h = NULL;
  units_n_send_cgps_d = NULL;
  n_recv_cgps = 0;
  n_send_cgps = 0;
  recv_cgp_mem_h = NULL;
  recv_cgp_mem_d = NULL;
  send_cgp_mem_h = NULL;
  send_cgp_mem_d = NULL;
  recv_cgp_start_h = NULL;
  recv_cgp_start_d = NULL;
  send_cgp_start_h = NULL;
  send_cgp_start_d = NULL;
  recv_cons_cnt = 0;
  send_cons_cnt = 0;
  recv_cons_mem_h = NULL;
  recv_cons_mem_d = NULL;
  send_cons_mem_h = NULL;
  send_cons_mem_d = NULL;
  
  // own_cons_max_size = 0;
  // own_cons_tot_size = 0;
  // own_cons_tot_size_nonshared = 0;
  // own_cons_avg_size = 0;
  // own_cons_max_vars = 0;

  n_unit_specs = 0;
  unit_spec_size = 0;
  unit_spec_mem_tot = 0;
  unit_spec_mem_h = NULL;
  unit_spec_mem_d = NULL;
  n_con_specs = 0;
  con_spec_size = 0;
  con_spec_mem_tot = 0;
  con_spec_mem_h = NULL;
  con_spec_mem_d = NULL;
}

void Network_cuda::NetFree() {
  if(recv_cons_mem_d)
    hipFree(recv_cons_mem_d);
  if(send_cons_mem_d)
    hipFree(send_cons_mem_d);

  if(recv_cgp_mem_d)
    hipFree(recv_cgp_mem_d);
  if(send_cgp_mem_d)
    hipFree(send_cgp_mem_d);
  if(recv_cgp_start_d)
    hipFree(recv_cgp_start_d);
  if(send_cgp_start_d)
    hipFree(send_cgp_start_d);
  
  if(units_n_recv_cgps_d)
    hipFree(units_n_recv_cgps_d);
  if(units_n_send_cgps_d)
    hipFree(units_n_send_cgps_d);

  if(lay_stats_d)
    hipFree(lay_stats_d);
  
  if(units_mem_d)
    hipFree(units_mem_d);

  if(unit_spec_mem_h)
    free(unit_spec_mem_h);
  if(unit_spec_mem_d)
    hipFree(unit_spec_mem_d);

  if(con_spec_mem_h)
    free(con_spec_mem_h);
  if(con_spec_mem_d)
    hipFree(con_spec_mem_d);

  if(strms_created) {
    hipStreamDestroy(strm_memcpy_cons);
    hipStreamDestroy(strm_memcpy_units);
    hipStreamDestroy(strm_compute_netin);
    hipStreamDestroy(strm_compute_dwt);
    hipStreamDestroy(strm_compute_wt);
    strms_created = false;
  }

  Initialize();
}

void Network_cuda::NetAlloc
( int    uvs,
  int    nub,
  int    nlb,
  int    nugb,
  char*  umh,
  int*   luih,
  int*   uuih,

  int    nls,
  int    nlsv,
  float* lsh,

  bool   roc,
  int*   unrch,
  int*   unsch,
  int    nrcg,
  int    nscg,
  int*   rcsh,
  int*   scsh,

  cudabigint rcc,
  cudabigint scc,
  float* rcmh,
  float* scmh
  )
{
  if(!strms_created) {
    hipStreamCreate(&strm_memcpy_cons);
    hipStreamCreate(&strm_memcpy_units);
    hipStreamCreate(&strm_compute_netin);
    hipStreamCreate(&strm_compute_act);
    hipStreamCreate(&strm_compute_dwt);
    hipStreamCreate(&strm_compute_wt);
    strms_created = true;
  }
  
  unit_vars_size = uvs;
  n_units_built = nub;
  n_layers_built = nlb;
  n_ungps_built = nugb;

  units_mem_h = umh;
  cudaSafeCall(hipMalloc(&units_mem_d, n_units_built * unit_vars_size));

  lay_unit_idxs_h = luih;
  cudaSafeCall(hipMalloc(&lay_unit_idxs_d, n_layers_built * 2 * sizeof(int)));
  ungp_unit_idxs_h = uuih;
  cudaSafeCall(hipMalloc(&ungp_unit_idxs_d, n_ungps_built * 2 * sizeof(int)));
  
  n_lay_stats = nls;
  n_lay_stats_vars = nlsv;
  lay_stats_h = lsh;
  cudaSafeCall(hipMalloc(&lay_stats_d, n_lay_stats * n_lay_stats_vars * n_layers_built * sizeof(float)));

  recv_owns_cons = roc;
  
  units_n_recv_cgps_h = unrch;
  cudaSafeCall(hipMalloc(&units_n_recv_cgps_d, n_units_built * sizeof(int)));
  units_n_send_cgps_h = unsch;
  cudaSafeCall(hipMalloc(&units_n_send_cgps_d, n_units_built * sizeof(int)));

  n_recv_cgps = nrcg;
  n_send_cgps = nscg;

  con_group_size = sizeof(ConGroup_cuda);
  recv_cgp_mem_h = (char*)malloc(n_recv_cgps * con_group_size);
  cudaSafeCall(hipMalloc(&recv_cgp_mem_d, n_recv_cgps * con_group_size));
  send_cgp_mem_h = (char*)malloc(n_send_cgps * con_group_size);
  cudaSafeCall(hipMalloc(&send_cgp_mem_d, n_send_cgps * con_group_size));
  
  recv_cgp_start_h = rcsh;
  cudaSafeCall(hipMalloc(&recv_cgp_start_d, n_units_built * sizeof(int)));
  send_cgp_start_h = scsh;
  cudaSafeCall(hipMalloc(&send_cgp_start_d, n_units_built * sizeof(int)));

  recv_cons_cnt = rcc;
  send_cons_cnt = scc;
  
  recv_cons_mem_h = rcmh;
  cudaSafeCall(hipMalloc(&recv_cons_mem_d, recv_cons_cnt * sizeof(float)));
  send_cons_mem_h = scmh;
  cudaSafeCall(hipMalloc(&send_cons_mem_d, send_cons_cnt * sizeof(float)));
}

void Network_cuda::NetToDevice() {
  if(!lay_unit_idxs_h) return;

  cudaSafeCall
    (hipMemcpy(lay_unit_idxs_d, lay_unit_idxs_h, n_layers_built * 2 * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(ungp_unit_idxs_d, ungp_unit_idxs_h, n_ungps_built * 2 * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(units_n_recv_cgps_d, units_n_recv_cgps_h, n_units_built * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(units_n_send_cgps_d, units_n_send_cgps_h, n_units_built * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(recv_cgp_start_d, recv_cgp_start_h, n_units_built * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(send_cgp_start_d, send_cgp_start_h, n_units_built * sizeof(int),
                hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(recv_cgp_mem_d, recv_cgp_mem_h, n_recv_cgps * con_group_size,
                          hipMemcpyHostToDevice));
  cudaSafeCall
    (hipMemcpy(send_cgp_mem_d, send_cgp_mem_h, n_send_cgps * con_group_size,
                          hipMemcpyHostToDevice));

  if(!recv_owns_cons) {         // copy the NON-owned cons -- just structural -- no data
    cudaSafeCall
      (hipMemcpy(recv_cons_mem_d, recv_cons_mem_h, recv_cons_cnt * sizeof(float),
                  hipMemcpyHostToDevice));
  }
  else {
    cudaSafeCall
      (hipMemcpy(send_cons_mem_d, send_cons_mem_h, send_cons_cnt * sizeof(float),
                  hipMemcpyHostToDevice));
  }
}

int Network_cuda::SetCudaParams(int min_th, int max_th, int cons_per_th,
                                int own_cons_avg_size) {
  min_threads = min_th;
  max_threads = max_th;
  cons_per_thread = cons_per_th;

  // docs on number of threads: http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#execution-configuration-optimizations

  n_threads = (int)round((float)own_cons_avg_size / (float)cons_per_thread);
  int mod32 = n_threads % 32;
  if(mod32 > 15)                // round up
    n_threads = ((n_threads / 32) + 1) * 32;
  else
    n_threads = (n_threads / 32) * 32;
  if(n_threads < min_threads)
    n_threads = min_threads;
  if(n_threads > max_threads)
    n_threads = max_threads;
  return n_threads;
}

void Network_cuda::OwnCons_HostToDevice(bool sync) {
  if(recv_owns_cons) {
    if(!(recv_cons_mem_h && recv_cons_mem_d)) return;
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_d, recv_cons_mem_h,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyHostToDevice, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
  else {
    if(!(send_cons_mem_h && send_cons_mem_d)) return;
    cudaSafeCall(hipMemcpyAsync(send_cons_mem_d, send_cons_mem_h,
                                 send_cons_cnt * sizeof(float),
                                 hipMemcpyHostToDevice, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::OwnCons_DeviceToHost(bool sync) {
  if(recv_owns_cons) {
    if(!(recv_cons_mem_h && recv_cons_mem_d)) return;
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_h, recv_cons_mem_d,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyDeviceToHost, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
  else {
    if(!(send_cons_mem_h && send_cons_mem_d)) return;
    cudaSafeCall(hipMemcpyAsync(send_cons_mem_h, send_cons_mem_d,
                                 send_cons_cnt * sizeof(float),
                                 hipMemcpyDeviceToHost, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::UnitVars_HostToDevice(bool sync) {
  if(!(units_mem_h && units_mem_d)) return;
  cudaSafeCall
    (hipMemcpyAsync(units_mem_d, units_mem_h, n_units_built * unit_vars_size,
                     hipMemcpyHostToDevice, strm_memcpy_units));
  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_memcpy_units));
  }
}

void Network_cuda::UnitVars_DeviceToHost(bool sync) {
  if(!(units_mem_h && units_mem_d)) return;
  cudaSafeCall
    (hipMemcpyAsync(units_mem_h, units_mem_d, n_units_built * unit_vars_size,
                     hipMemcpyDeviceToHost, strm_memcpy_units));
  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_memcpy_units));
  }
}

void Network_cuda::ExtInputToDevice(bool sync) {
  if(!(units_mem_h && units_mem_d)) return;
  for(int li=0; li < n_layers_built; li++) {
    int st_ui = LayUnStart(lay_unit_idxs_h, li);
    int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    int nu = ed_ui - st_ui;

    UnitVars_cuda* u = (UnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem_h, unit_vars_size, st_ui);
    
    if(!((u->ext_flag & UnitVars_cuda::EXT) || (u->ext_flag & UnitVars_cuda::TARG)))
      continue;
    
    cudaSafeCall
      (hipMemcpyAsync(units_mem_d + st_ui * unit_vars_size,
                       units_mem_h + st_ui * unit_vars_size,
                       nu * unit_vars_size,
                       hipMemcpyHostToDevice, strm_memcpy_units));
  }
  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_memcpy_units));
  }
}

void Network_cuda::TargUnitsToHost(bool sync) {
  if(!(units_mem_h && units_mem_d)) return;
  for(int li=0; li < n_layers_built; li++) {
    int st_ui = LayUnStart(lay_unit_idxs_h, li);
    int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    int nu = ed_ui - st_ui;

    UnitVars_cuda* u = (UnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem_h, unit_vars_size, st_ui);
    
    if(!(u->ext_flag & UnitVars_cuda::TARG))
      continue;
    
    cudaSafeCall
      (hipMemcpyAsync(units_mem_h + st_ui * unit_vars_size,
                       units_mem_d + st_ui * unit_vars_size,
                       nu * unit_vars_size,
                       hipMemcpyDeviceToHost, strm_memcpy_units));
  }
  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_memcpy_units));
  }
}

bool Network_cuda::AllocUnitSpecs(int n_us) {
  if(unit_spec_mem_h)
    free(unit_spec_mem_h);
  if(unit_spec_mem_d)
    hipFree(unit_spec_mem_d);

  n_unit_specs = n_us;
  if(n_us == 0)                 // can't really happen..
    return false;

  unit_spec_mem_tot = n_unit_specs * unit_spec_size;
  if(unit_spec_mem_tot > max_constant_mem)
    return false;
  
  unit_spec_mem_h = (char*)malloc(unit_spec_mem_tot);
  cudaSafeCall(hipMalloc(&unit_spec_mem_d, n_unit_specs * unit_spec_size));
  return true;
}

void Network_cuda::UnitSpecs_HostToDevice() {
  if(!(unit_spec_mem_h && unit_spec_mem_d)) return;
  
  cudaSafeCall
    (hipMemcpy(unit_spec_mem_d, unit_spec_mem_h, n_unit_specs * unit_spec_size,
                hipMemcpyDeviceToHost));
}

bool Network_cuda::AllocConSpecs(int n_us) {
  if(con_spec_mem_h)
    free(con_spec_mem_h);
  if(con_spec_mem_d)
    hipFree(con_spec_mem_d);

  n_con_specs = n_us;
  if(n_us == 0)                 // can't really happen..
    return false;

  con_spec_mem_tot = n_con_specs * con_spec_size;
  if(con_spec_mem_tot > max_constant_mem)
    return false;
  
  con_spec_mem_h = (char*)malloc(con_spec_mem_tot);
  cudaSafeCall(hipMalloc(&con_spec_mem_d, n_con_specs * con_spec_size));
  return true;
}

void Network_cuda::ConSpecs_HostToDevice() {
  if(!(con_spec_mem_h && con_spec_mem_d)) return;
  
  cudaSafeCall
    (hipMemcpy(con_spec_mem_d, con_spec_mem_h, n_con_specs * con_spec_size,
                hipMemcpyDeviceToHost));
}


