#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "Network_cuda.h"

//  lldb print commands -- use memory read with format
//  me r -rf f -l148 -c148 send_netin_tmp_h

Network_cuda::Network_cuda() {
  Initialize();
}

Network_cuda::~Network_cuda() {
  FreeCudaArrays();
}

void Network_cuda::Initialize() {
  strms_created = false;

  con_group_size = 0;
  unit_vars_size = 0;
  n_units_built = 0;
  n_layers_built = 0;
  n_ungps_built = 0;
  units_mem_h = NULL;
  units_mem_d = NULL;
  n_lay_stats = 0;
  n_lay_stats_vars = 0;
  lay_stats_h = NULL;
  lay_stats_d = NULL;
  units_n_recv_cgps_h = NULL;
  units_n_recv_cgps_d = NULL;
  units_n_send_cgps_h = NULL;
  units_n_send_cgps_d = NULL;
  n_recv_cgps = 0;
  n_send_cgps = 0;
  recv_cgp_mem_h = NULL;
  recv_cgp_mem_d = NULL;
  send_cgp_mem_h = NULL;
  send_cgp_mem_d = NULL;
  recv_cgp_start_h = NULL;
  recv_cgp_start_d = NULL;
  send_cgp_start_h = NULL;
  send_cgp_start_d = NULL;
  recv_cons_cnt = 0;
  send_cons_cnt = 0;
  recv_cons_mem_h = NULL;
  recv_cons_mem_d = NULL;
  send_cons_mem_h = NULL;
  send_cons_mem_d = NULL;
  own_cons_max_size = 0;
  own_cons_tot_size = 0;
  own_cons_tot_size_nonshared = 0;
  own_cons_avg_size = 0;
  own_cons_max_vars;

  con_params_h = NULL;
  con_params_d = NULL;
}

void Network_cuda::FreeCudaArrays() {
  if(recv_cons_mem_d)
    hipFree(recv_cons_mem_d);
  if(send_cons_mem_d)
    hipFree(send_cons_mem_d);

  if(recv_cgp_mem_d)
    hipFree(recv_cgp_mem_d);
  if(send_cgp_mem_d)
    hipFree(send_cgp_mem_d);
  if(recv_cgp_start_mem_d)
    hipFree(recv_cgp_start_mem_d);
  if(send_cgp_start_mem_d)
    hipFree(send_cgp_start_mem_d);
  
  if(units_n_recv_cgps_d)
    hipFree(units_n_recv_cgps_d);
  if(units_n_send_cgps_d)
    hipFree(units_n_send_cgps_d);

  if(lay_stats_d)
    hipFree(lay_stats_d);
  
  if(units_mem_d)
    hipFree(units_mem_d);

  if(con_params_h)
    free(con_params_h);
  if(con_params_d)
    hipFree(con_params_d);

   if(strms_created) {
    hipStreamDestroy(strm_memcpy_cons);
    hipStreamDestroy(strm_memcpy_units);
    hipStreamDestroy(strm_compute_netin);
    hipStreamDestroy(strm_compute_dwt);
    hipStreamDestroy(strm_compute_wt);
    strms_created = false;
  }

  Initialize();
}

void Network_cuda::AllocCudaArrays
( int    cgs,
  int    uvs,
  int    nub,
  int    nlb,
  int    nugb,
  char*  umh,
  int*   luih,
  int*   uuih,

  int    nls,
  int    nlsv,
  float* lsh,
  int*   unrch,
  int*   unsch,
  
  int    nrcg,
  int    nscg,

  char*  rcgmh,
  char*  scgmh,
  int*   rcsh,
  int*   scsh,

  bigint rcc,
  bigint scc,
  float* rcmh,
  float* scmh
  )
{
  if(!strms_created) {
    hipStreamCreate(&strm_memcpy_cons);
    hipStreamCreate(&strm_memcpy_units);
    hipStreamCreate(&strm_compute_netin);
    hipStreamCreate(&strm_compute_act);
    hipStreamCreate(&strm_compute_dwt);
    hipStreamCreate(&strm_compute_wt);
    strms_created = true;
  }
  
  min_threads = min_th;
  max_threads = max_th;
  cons_per_thread = cons_per_th;

  // docs on number of threads: http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#execution-configuration-optimizations

  n_threads = (int)round((float)own_cons_avg_size / (float)cons_per_thread);
  int mod32 = n_threads % 32;
  if(mod32 > 15)                // round up
    n_threads = ((n_threads / 32) + 1) * 32;
  else
    n_threads = (n_threads / 32) * 32;
  if(n_threads < min_threads)
    n_threads = min_threads;
  if(n_threads > max_threads)
    n_threads = max_threads;

  con_group_size = cgs;
  unit_vars_size = uvs;
  n_units_built = nub;
  n_layers_built = nlb;
  n_ungps_built = nugb;

  units_mem_h = umh;
  cudaSafeCall(hipMalloc(&units_mem_d, n_units_built * sizeof(char)));

  lay_unit_idxs_h = luih;
  cudaSafeCall(hipMalloc(&lay_unit_idxs_d, n_layers_built * 2 * sizeof(int)));
  ungp_unit_idxs_h = uuih;
  cudaSafeCall(hipMalloc(&ungp_unit_idxs_d, n_ungps_built * 2 * sizeof(int)));
  
  n_lay_stats = nls;
  n_lay_stats_vars = nlsv;
  lay_stats_h = lsh;
  cudaSafeCall(hipMalloc(&lay_stats_d, n_lay_stats * n_lay_stats_vars * n_layers_built * sizeof(float)));

  units_n_recv_cgps_h = unrch;
  cudaSafeCall(hipMalloc(&units_n_recv_cgps_d, n_units_built * sizeof(int)));
  units_n_send_cgps_h = unsch;
  cudaSafeCall(hipMalloc(&units_n_send_cgps_d, n_units_built * sizeof(int)));

  n_recv_cgps = nrcg;
  n_send_cgps = nscg;

  recv_cgp_mem_h = rcgmh;
  cudaSafeCall(hipMalloc(&recv_cgp_mem_d, n_recv_cgps * con_group_size * sizeof(char)));
  send_cgp_mem_h = scgmh;
  cudaSafeCall(hipMalloc(&send_cgp_mem_d, n_send_cgps * con_group_size * sizeof(char)));
  
  recv_cgp_start_h = rcsh;
  cudaSafeCall(hipMalloc(&recv_cgp_start_d, n_units_built * sizeof(int)));
  send_cgp_start_h = scsh;
  cudaSafeCall(hipMalloc(&send_cgp_start_d, n_units_built * sizeof(int)));

  recv_cons_cnt = rcc;
  send_cons_cnt = scc;
  
  recv_cons_mem_h = rcmh;
  cudaSafeCall(hipMalloc(&recv_cons_mem_d, recv_cons_cnt * sizeof(float)));
  send_cons_mem_h = scmh;
  cudaSafeCall(hipMalloc(&send_cons_mem_d, send_cons_cnt * sizeof(float)));
  
  con_params_h = (float*)malloc(n_recv_cgps * N_CON_PARAMS * sizeof(float));
  cudaSafeCall(hipMalloc(&con_params_d, n_recv_cgps * N_CON_PARAMS * sizeof(float)));
}

void Network_cuda::RecvCons_HostToDevice(bool sync) {
  if(recv_cons_mem_h && recv_coms_mem_d) {
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_d, recv_cons_mem_h,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyHostToDevice, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::RecvCons_DeviceToHost(bool sync) {
  if(recv_cons_mem_h && recv_cons_mem_d) {
    cudaSafeCall(hipMemcpyAsync(recv_cons_mem_h, recv_cons_mem_d,
                                 recv_cons_cnt * sizeof(float),
                                 hipMemcpyDeviceToHost, strm_memcpy_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_memcpy_cons));
    }
  }
}

void Network_cuda::UpdateUnitsXCons() {
  if(!con_mem_idxs_h) return;

  int sz =  recv_units_x_cons * sizeof(int);

  cudaSafeCall(hipMemcpy(units_d, units_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_mem_idxs_d, con_mem_idxs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_allocs_d, con_allocs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_sizes_d, con_sizes_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_recv_idxs_d, con_recv_idxs_h, sz, hipMemcpyHostToDevice));
}

void Network_cuda::UpdateConParams() {
  if(!con_params_h) return;

  int sz =  own_units_x_cons * N_CON_PARAMS * sizeof(float);

  cudaSafeCall(hipMemcpy(con_params_d, con_params_h, sz, hipMemcpyHostToDevice));

  if(wt_sig_fun_h) {
    cudaSafeCall(hipMemcpy(wt_sig_fun_d, wt_sig_fun_h, 10002 * sizeof(float),
                            hipMemcpyHostToDevice));
  }
}

// todo: definitely should just not use the entire congroup structure
// and instead just have a size int per congroup and a temp float array for agg results


__global__ void Kernel_Compute_Netin_OneLayer
(const int cgp_st_idx, char* net_cgp_mem, const int con_group_size, char* net_recv_cons_mem, char* net_units_mem, const int unit_vars_size) {

  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int cgp_idx = blockIdx.x + cgp_st_idx;

  __shared__ float temp_sums[nthrs];

  ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(net_cgp_mem, con_group_size, cgp_idx);

  const int sz = cg->size;
  
  const float cn_per_th = ((float)sz / (float)nthrs);
  int st = __float2int_rn((float)thr_no * cn_per_th);
  int ed = __float2int_rn((float)(thr_no+1) * cn_per_th);
  ed = ed < sz ? ed : sz;     // max of sz
  // Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  const float* wts = cg->OwnCnVar(net_recv_cons_mem, ConGroup_cuda::WT);

  float sum = 0.0f;

  while(st < ed) {
    const int32_t su_idx = cg->UnIdx(net_recv_cons_mem, st);
    UnitVars_cuda* su = Network_cuda::GetUnitVars(net_units_mem, unit_vars_size, unit_idx);
    sum += wts[st] * su->act;
    st++;
  }

  temp_sums[thr_no] = sum;

  __synchthreads();            // make sure all threads have written to temp_sums

  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i)
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up

    __synchthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    // super cheesy: using otherwise unused mem_start memory to cache the netin result
    ((float)(cg->mem_start) = temp_sums[0]; // first guy has it all
  }
}

void Network_cuda::Compute_NetinAct() {
  for(int i=0; i< n_layers_built; i++) {
    int st_ui = LayUnStart(lay_unit_idxs_h, i);
    int ed_ui = LayUnEnd(lay_unit_idxs_h, i);
    int nu = ed_ui - st_ui;

    int cgp_st_idx = recv_cgp_start_h[st_ui];
    
    //  Invoke kernel
    Kernel_Compute_Netin_OneLayer<<<nu, n_threads, 0, strm_compute_netin>>>
      (cgp_st_idx, recv_cgp_mem_d, con_group_size, recv_cons_mem_d, units_mem_d,
       unit_vars_size);
    cudaSafeCall(hipStreamSynchronize(strm_compute_netin));

    // then aggregate netins and compute activations, all in a thread??
    
  }
}

// __global__ void Kernel_Compute_dWt
// (int* cur_units_x_cons_d, float* unit_vec_vars_d, float* con_params_d, int* units_d,
//  float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
//  const int nu) {
//   const int csni = blockIdx.x;
//   const int nth = blockDim.x;
//   const int ucidx = cur_units_x_cons_d[csni];
//   const int sidx = units_d[ucidx];

//   const float su_avg_s = unit_vec_vars_d[Network_cuda::AVG_S * nu + sidx];
//   const float su_avg_m = unit_vec_vars_d[Network_cuda::AVG_M * nu + sidx];

//   const float s_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
//                                    Network_cuda::S_MIX];
//   const float m_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
//                                    Network_cuda::M_MIX];
//   const float thr_l_mix = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
//                                        Network_cuda::THR_L_MIX];
//   const float thr_max = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
//                                      Network_cuda::THR_MAX];
//   const float clrate = con_params_d[ucidx * Network_cuda::N_CON_PARAMS +
//                                     Network_cuda::CUR_LRATE];

//   const int sz = con_sizes_d[ucidx];
//   float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
//     (con_allocs_d[ucidx] * (1 + Network_cuda::DWT));
//   const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
//   int th = threadIdx.x;
//   const float cn_per_th = ((float)sz / (float)nth);
//   int st = __float2int_rn((float)th * cn_per_th);
//   int ed = __float2int_rn((float)(th+1) * cn_per_th);
//   //  ed = ed < sz ? ed : sz;     // max of sz
//   while(st < ed) {
//     int ridx = ridxs[st];
//     const float ru_avg_s = unit_vec_vars_d[Network_cuda::AVG_S * nu + ridx];
//     const float ru_avg_m = unit_vec_vars_d[Network_cuda::AVG_M * nu + ridx];
//     const float ru_avg_l = unit_vec_vars_d[Network_cuda::AVG_L * nu + ridx];

//     // unfortunately, cos_diff_lmix is on recv layer -- so this needs to be in the loop
//     // whereas normally it is outside the loop.. would require a separate
//     // var array just for it, at the unit_x_con level..  could look into it later
//     const float cos_diff_lmix = unit_vec_vars_d[Network_cuda::COS_DIFF_LMIX * nu
//                                                 + ridx];
//     const float efflmix = thr_l_mix * cos_diff_lmix;
//     const float effmmix = 1.0f - efflmix;
//     const float su_act_mult = efflmix * su_avg_m;

//     const float srs = ru_avg_s * su_avg_s;
//     const float srm = ru_avg_m * su_avg_m;
//     const float sm_mix = s_mix * srs + m_mix * srm;
//     const float lthr = su_act_mult * ru_avg_l;
//     float effthr = effmmix * srm + lthr;
//     effthr = effthr < thr_max ? effthr : thr_max; // max = thr_max

//     float rval;                 // xcal.dWtFun
//     if(sm_mix < 0.0001f)        // d_thr = 0.0001
//       rval = 0.0f;
//     else if(sm_mix > effthr * 0.1f) // d_rev = 0.1
//       rval = (sm_mix - effthr);
//     else
//       rval = sm_mix * -9.0f;    // d_rev_ratio = -9.0;
//     dwts[st] += clrate * rval;
//     st++;
//   }
// }

// void Network_cuda::Compute_dWt(bool sync) {
//   if(cur_units_x_cons_n == 0) return;

//   cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
//                                cur_units_x_cons_n * sizeof(int),
//                                hipMemcpyHostToDevice, strm_compute_dwt));
//   cudaSafeCall(hipMemcpyAsync(unit_vec_vars_d, unit_vec_vars_h,
//                                (n_units+1) * N_VEC_VARS * sizeof(float),
//                                hipMemcpyHostToDevice, strm_compute_dwt));

//   //  Invoke kernel
//   Kernel_Compute_dWt_cosdif<<<cur_units_x_cons_n, n_threads, 0, strm_compute_dwt>>>
//     (cur_units_x_cons_d, unit_vec_vars_d, con_params_d, units_d,
//      own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, n_units+1);

//   if(sync) {
//     cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
//   }
// }

// __global__ void Kernel_Compute_Weights
// (float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
//  float* wt_sig_fun_d) {
//   const int ucidx = blockIdx.x;  // full unit x con idx here
//   const int nth = blockDim.x;

//   const int sz = con_sizes_d[ucidx];
//   float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
//     (con_allocs_d[ucidx] * (1 + Network_cuda::WT));
//   float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
//     (con_allocs_d[ucidx] * (1 + Network_cuda::DWT));
//   float* fwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
//     (con_allocs_d[ucidx] * (1 + Network_cuda::FWT));
//   float* swts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
//     (con_allocs_d[ucidx] * (1 + Network_cuda::SWT));

//   int th = threadIdx.x;
//   const float cn_per_th = ((float)sz / (float)nth);
//   int st = __float2int_rn((float)th * cn_per_th);
//   int ed = __float2int_rn((float)(th+1) * cn_per_th);
//   //  ed = ed < sz ? ed : sz;     // max of sz
//   while(st < ed) {
//     float& dwt = dwts[st];
//     if(dwt != 0.0f) {
//       float& wt = wts[st];
//       float& fwt = fwts[st];
//       float& swt = swts[st];
//       if(dwt > 0.0f)  dwt *= (1.0f - fwt);
//       else            dwt *= fwt;
//       fwt += dwt;
//       swt = fwt;                // keep sync'd -- not tech necc..

//       int idx = __float2int_rd(fwt * 10000.0f); // sig_res_inv
//       wt = wt_sig_fun_d[idx];

//       dwt = 0.0f;
//     }
//     st++;
//   }
// }

// void Network_cuda::Compute_Weights(bool sync) {
//   //  Invoke kernel -- does all
//   Kernel_Compute_Weights<<<own_units_x_cons, n_threads, 0, strm_compute_wt>>>
//     (own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, wt_sig_fun_d);

//   if(sync) {
//     cudaSafeCall(hipStreamSynchronize(strm_compute_wt));
//   }
// }



