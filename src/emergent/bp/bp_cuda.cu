#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "bp_cuda.h"

/////////////////////////////////////////////
//      layer-at-a-time netinput and act

__global__ void Kernel_Compute_Netin_OneLayer
(const int cgp_st_idx, char* net_cgp_mem, const int con_group_size, float* net_recv_cons_mem, char* net_units_mem, const int unit_vars_size) {

  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int cgp_idx = blockIdx.x + cgp_st_idx;

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(net_cgp_mem, con_group_size, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  const float* wts = cg->OwnCnVar(net_recv_cons_mem, ConGroup_cuda::WT);

  float sum = 0.0f;

  while(st < ed) {
    const int32_t su_idx = cg->UnIdx(net_recv_cons_mem, st);
    UnitVars_cuda* su = Network_cuda::GetUnitVars(net_units_mem, unit_vars_size, su_idx);
    sum += wts[st] * su->act;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i)
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up

    __syncthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  }
}

// this is just sample code showing how to roll up the netin and then compute act from
// that -- actual algorithm will have to do this with its own special code for act fun

__global__ void Kernel_Compute_Act_OneLayer
(const int un_st_idx, char* units_mem, const int unit_vars_size,
 int* recv_cgp_start, int* units_n_recv_cgps,
 char* net_cgp_mem, const int con_group_size) {

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = un_st_idx + blockIdx.x * nthrs + thr_no; // un_st_idx = 1 at minimum..

  UnitVars_cuda* u = Network_cuda::GetUnitVars(units_mem, unit_vars_size, un_idx);

  if(!(u->ext_flag & UnitVars_cuda::EXT)) {
    // first step is to sum up the netins for this guy
    float netin = 0.0f;
    const int cgp_st_idx = recv_cgp_start[un_idx];
    if(cgp_st_idx > 0) {
      const int n_cgps = units_n_recv_cgps[un_idx];
      for(int i=0; i<n_cgps; i++) {
        ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(net_cgp_mem, con_group_size,
                                                           cgp_st_idx + i);
        netin += cg->temp1;
      }
    }

    u->net = netin;
    // exponential activation example:
    u->act = 1.0f / (1.0f + expf(netin));
  }
  else {
    u->act = u->ext;
  }
}

void Bp_cuda::Compute_NetinAct() {
  ExtInputToDevice(true);       // typically need the exts first!
  
  for(int i=0; i< n_layers_built; i++) {
    int st_ui = LayUnStart(lay_unit_idxs_h, i);
    int ed_ui = LayUnEnd(lay_unit_idxs_h, i);
    int nu = ed_ui - st_ui;

    int cgp_st_idx = recv_cgp_start_h[st_ui];
    if(cgp_st_idx > 0) {
      //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
      Kernel_Compute_Netin_OneLayer<<<nu, n_threads, n_threads, strm_compute_netin>>>
        (cgp_st_idx, recv_cgp_mem_d, con_group_size, recv_cons_mem_d, units_mem_d,
         unit_vars_size);
    }
    cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
    // then aggregate netins and compute activations
    Kernel_Compute_Act_OneLayer<<<nu, n_threads, 0, strm_compute_netin>>>
      (st_ui, units_mem_d, unit_vars_size, recv_cgp_start_d, units_n_recv_cgps_d,
       recv_cgp_mem_d, con_group_size);
    cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
  }
}


