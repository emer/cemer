#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "bp_cuda.h"

Bp_cuda::Bp_cuda() {
  unit_spec_size = sizeof(BpUnitSpec_cuda);
  con_spec_size = sizeof(BpConSpec_cuda);
}

// this constant memory is used for holding the spec info:

__constant__ char const_spec_mem[Network_cuda::max_constant_mem];


///////////////////////////////////////////////////////////////////////////
//      layer-at-a-time netinput and act

__global__ void Kernel_Compute_Netin_ConGroup
(const int st_ui, const int cgp_idx, int* recv_cgp_start, char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, char* units_mem, const int unit_vars_size) {

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  ConGroup_cuda* cg = Network_cuda::GetUnConGroup
    (recv_cgp_mem, recv_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  const float* wts = cg->OwnCnVar(recv_cons_mem, ConGroup_cuda::WT);

  float sum = 0.0f;

  while(st < ed) {
    const int32_t su_idx = cg->UnIdx(recv_cons_mem, st);
    UnitVars_cuda* su = Network_cuda::GetUnitVars(units_mem, unit_vars_size, su_idx);
    sum += wts[st] * su->act;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i) {
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
    }
    __syncthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  }
}


__global__ void Kernel_Compute_BpAct
(const int un_st_idx, char* units_mem, const int unit_vars_size, int unit_spec_size,
 int* recv_cgp_start, int* units_n_recv_cgps, char* net_cgp_mem, const int con_group_size)
{

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = un_st_idx + blockIdx.x * nthrs + thr_no; // un_st_idx = 1 at minimum..

  UnitVars_cuda* u = Network_cuda::GetUnitVars(units_mem, unit_vars_size, un_idx);

  if(!(u->ext_flag & UnitVars_cuda::EXT)) {
    // first step is to sum up the netins for this guy
    float netin = 0.0f;
    const int cgp_st_idx = recv_cgp_start[un_idx];
    if(cgp_st_idx > 0) {
      const int n_cgps = units_n_recv_cgps[un_idx];
      for(int i=0; i<n_cgps; i++) {
        ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(net_cgp_mem, con_group_size,
                                                           cgp_st_idx + i);
        netin += cg->temp1;
      }
    }

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);
    
    u->net = netin;
    u->act = us->ActFromNetin(netin);
    // todo: dropout.. noise..
  }
  else {
    u->act = u->ext;
  }
}

void Bp_cuda::Compute_NetinAct() {
  ExtInputToDevice(true);       // external input comes from host..
  
  const int nlay = n_layers_built;
  for(int li=0; li < nlay; li++) {
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    int n_cgps = units_n_recv_cgps_h[st_ui]; // assume same for all..
    if(n_cgps > 0) {
      for(int cgi = 0; cgi < n_cgps; cgi++) {
        //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
        Kernel_Compute_Netin_ConGroup<<<nu, n_threads, n_threads, strm_compute_netin>>>
          (st_ui, cgi, recv_cgp_start_d, recv_cgp_mem_d, con_group_size,
           recv_cons_mem_d, units_mem_d, unit_vars_size);
        cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
      }
      // then aggregate netins and compute activations
    }

    // copy unit spec mem to constant
    hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
    
    Kernel_Compute_BpAct<<<nu, n_threads, 0, strm_compute_act>>>
      (st_ui, units_mem_d, unit_vars_size, unit_spec_size,
       recv_cgp_start_d, units_n_recv_cgps_d, recv_cgp_mem_d, con_group_size);
    cudaSafeCall(hipStreamSynchronize(strm_compute_act));
  }
}


//////////////////////////////////////////////////////////////////////////
//      layer-at-a-time dEdA and dEdNet


__global__ void Kernel_Compute_dEdA_ConGroup
(const int st_ui, const int cgp_idx, int* send_cgp_start, char* send_cgp_mem,
 const int con_group_size, float* send_cons_mem, char* units_mem, const int unit_vars_size,
 int* recv_cgp_start, float* recv_cons_mem)
{

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  ConGroup_cuda* cg = Network_cuda::GetUnConGroup
    (send_cgp_mem, send_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float sum = 0.0f;

  while(st < ed) {
    const int32_t ru_idx = cg->UnIdx(send_cons_mem, st);
    BpUnitVars_cuda* ru = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem, unit_vars_size, ru_idx);
    // this is super deadly slow:
    const float wt = cg->PtrCn
      (send_cons_mem, send_cgp_mem, con_group_size, recv_cgp_start, st,
       recv_cons_mem, ConGroup_cuda::WT);
    sum += wt * ru->dEdNet;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i) {
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
    }
    __syncthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  }
}

__global__ void Kernel_Compute_Err_dEdNet
(const int un_st_idx, char* units_mem, const int unit_vars_size, int unit_spec_size,
 int* send_cgp_start, int* units_n_send_cgps, char* net_cgp_mem, const int con_group_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = un_st_idx + blockIdx.x * nthrs + thr_no; // un_st_idx = 1 at minimum..

  BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
    (units_mem, unit_vars_size, un_idx);

  // first step is to sum up the netins for this guy
  float dEdA = 0.0f;
  const int cgp_st_idx = send_cgp_start[un_idx];
  if(cgp_st_idx > 0) {
    const int n_cgps = units_n_send_cgps[un_idx];
    for(int i=0; i<n_cgps; i++) {
      ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(net_cgp_mem, con_group_size,
                                                         cgp_st_idx + i);
      dEdA += cg->temp1;
    }
  }

  BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
    (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);
    
  u->dEdA = dEdA;

  // compute error
  if(u->ext_flag & UnitVars_cuda::TARG) {
    float err = u->targ - u->act;
    if(fabsf(err) >= us->err_tol) {
      if(us->error_fun == BpUnitSpec_cuda::SQUARED_ERR
         || us->act_fun == BpUnitSpec_cuda::SOFTMAX) { // softmax always uses this!
        u->dEdA += err;
      }
      else { // CROSS_ENTROPY
        if(us->act_fun == BpUnitSpec_cuda::TANH) {
          err /= (u->act + 1.0f) * (1.0f - u->act) * 2.0f;
        }
        else {
          err /= u->act * (1.0f - u->act);
        }
        u->dEdA += err;
      }
    }
  }
    
  u->dEdNet = u->dEdA * us->ActDeriv(u->act);
}

void Bp_cuda::Compute_dEdA_dEdNet() {
  const int nlay = n_layers_built;
  for(int li = nlay-1; li >= 0; li--) { // go in reverse order!
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem_h, unit_vars_size, st_ui);
    if(u->ext_flag & UnitVars_cuda::EXT)
      continue;                 // skip it!  todo: could have bp to inputs but..

    int n_cgps = units_n_send_cgps_h[st_ui]; // assume same for all..
    if(n_cgps > 0) {
      for(int cgi = 0; cgi < n_cgps; cgi++) {
        //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
        Kernel_Compute_dEdA_ConGroup<<<nu, n_threads, n_threads, strm_compute_netin>>>
          (st_ui, cgi, send_cgp_start_d, send_cgp_mem_d, con_group_size,
           send_cons_mem_d, units_mem_d, unit_vars_size,
           recv_cgp_start_d, recv_cons_mem_d);
        cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
      }
      // then aggregate netins and compute activations
    }

    // copy unit spec mem to constant
    hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
    
    Kernel_Compute_Err_dEdNet<<<nu, n_threads, 0, strm_compute_act>>>
      (st_ui, units_mem_d, unit_vars_size, unit_spec_size,
       send_cgp_start_d, units_n_send_cgps_d, send_cgp_mem_d, con_group_size);
    cudaSafeCall(hipStreamSynchronize(strm_compute_act));
  }

  TargUnitsToHost(true);       // send output layer data back to host for stats..
}

