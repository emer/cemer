#include "hip/hip_runtime.h"
// Copyright 2017-2016, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "bp_cuda.h"

Bp_cuda::Bp_cuda() {
  unit_spec_size = sizeof(BpUnitSpec_cuda);
  con_spec_size = sizeof(BpConSpec_cuda);
}

// this constant memory is used for holding the spec info:

__constant__ char const_spec_mem[Network_cuda::max_constant_mem];


///////////////////////////////////////////////////////////////////////////
//      layer-at-a-time netinput and act

__global__ void Kernel_Compute_Netin_ConState
(const int st_ui, const int cgp_idx, int* recv_cgp_start, char* recv_cgp_mem,
 const int con_group_size, float* recv_cons_mem, char* units_mem, const int unit_vars_size) {

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetUnConState
    (recv_cgp_mem, recv_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  const float* wts = cg->OwnCnVar(recv_cons_mem, ConState_cuda::WT);

  float sum = 0.0f;

  while(st < ed) {
    UnitState_cuda* su = cg->UnVars(recv_cons_mem, units_mem, unit_vars_size, st);
    sum += wts[st] * su->act;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  // if(thr_no == 0) {
  //   sum = 0.0f;
  //   for(int i=0; i<nthrs; i++) {
  //     sum += temp_sums[i];
  //   }
  //   cg->temp1 = sum;
  // }
  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i) {
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
    }
    __syncthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  }
}


__global__ void Kernel_Compute_Act_Bp
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 int unit_spec_size, int* recv_cgp_start, int* units_n_recv_cgps, char* recv_cgp_mem,
 const int con_group_size)
{

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {

    UnitState_cuda* u = Network_cuda::GetUnitState(units_mem, unit_vars_size, un_idx);

    if(!(u->ext_flag & UnitState_cuda::EXT)) {
      // first step is to sum up the netins for this guy
      float netin = 0.0f;
      const int n_cgps = units_n_recv_cgps[un_idx];
      for(int i=0; i<n_cgps; i++) {
        ConState_cuda* cg = Network_cuda::GetUnConState
          (recv_cgp_mem, recv_cgp_start, con_group_size, un_idx, i);
        netin += cg->temp1;
      }

      netin += u->bias_wt;      // assume bias specs..
      
      BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
        (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);

      u->net = netin;
      u->act = us->ActFromNetin(netin);
      // todo: dropout.. noise..
    }
    else {
      u->act = u->ext;
    }
  }
}

void Bp_cuda::Compute_NetinAct() {
  ExtInputToDevice(true);       // external input comes from host..
  // IMPORTANT: this means that these layers MUST be sync'd *back to the host* for any
  // important changes, e.g., bias weights..
      
  // copy unit spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
  
  const int nlay = n_layers_built;
  for(int li=0; li < nlay; li++) {
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    if(nu > 0) {

      const int n_cgps = units_n_recv_cgps_h[st_ui]; // assume same for all..
      if(n_cgps > 0) {
        for(int cgi = 0; cgi < n_cgps; cgi++) {
          //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
          // use act stream here so netin and act are fully sync'd automatically
          Kernel_Compute_Netin_ConState<<<nu, n_threads, n_threads * sizeof(float), strm_compute_act>>>
            (st_ui, cgi, recv_cgp_start_d, recv_cgp_mem_d, con_group_size,
             recv_cons_mem_d, units_mem_d, unit_vars_size);
        }
      }

      // b/c units are accessed each per a diff thread, the n blocks is divided by threads
      const int n_blocks = (int)ceil((float)nu / (float)n_threads);

      Kernel_Compute_Act_Bp<<<n_blocks, n_threads, 0, strm_compute_act>>>
        (st_ui, ed_ui, units_mem_d, unit_vars_size, unit_spec_size,
         recv_cgp_start_d, units_n_recv_cgps_d, recv_cgp_mem_d, con_group_size);
      cudaSafeCall(hipStreamSynchronize(strm_compute_act));
    }
  }
}


//////////////////////////////////////////////////////////////////////////
//      layer-at-a-time dEdA and dEdNet


__global__ void Kernel_Compute_dEdA_ConState
(const int st_ui, const int cgp_idx, int* send_cgp_start, char* send_cgp_mem,
 const int con_group_size, float* send_cons_mem, char* units_mem, const int unit_vars_size,
 int* recv_cgp_start, char* recv_cgp_mem, float* recv_cons_mem)
{

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetUnConState
    (send_cgp_mem, send_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float sum = 0.0f;

  while(st < ed) {
    BpUnitState_cuda* ru = (BpUnitState_cuda*)cg->UnVars
      (send_cons_mem, units_mem, unit_vars_size, st);
    // this is super deadly slow:
    const float wt = cg->PtrCn
      (send_cons_mem, con_group_size, recv_cgp_start, recv_cgp_mem, recv_cons_mem, st,
       ConState_cuda::WT);
    sum += wt * ru->dEdNet;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  // if(thr_no == 0) {
  //   sum = 0;
  //   for(int i=0; i<nthrs; i++) {
  //     sum += temp_sums[i];
  //   }
  //   cg->temp1 = sum;
  // }
  int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  while( i!=0 ) {
    if(thr_no < i) {
      temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
    }
    __syncthreads();
    i /= 2;                     // binary tree -- only earlier and earlier threads get it
  }

  if(thr_no == 0) {
    cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  }
}

__global__ void Kernel_Compute_Err_dEdNet
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size, int unit_spec_size,
 int* send_cgp_start, int* units_n_send_cgps, char* send_cgp_mem, const int con_group_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;

  if(un_idx < ed_ui) {
    
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    // first step is to sum up the dEdA's for this guy
    float dEdA = 0.0f;
    const int n_cgps = units_n_send_cgps[un_idx];
    for(int i=0; i<n_cgps; i++) {
      ConState_cuda* cg = Network_cuda::GetUnConState
        (send_cgp_mem, send_cgp_start, con_group_size, un_idx, i);
      dEdA += cg->temp1;
    }

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);
    
    u->dEdA = dEdA;

    // compute error
    if(u->ext_flag & UnitState_cuda::TARG) {
      float err = u->targ - u->act;
      if(fabsf(err) >= us->err_tol) {
        if(us->error_fun == BpUnitSpec_cuda::SQUARED_ERR
           || us->act_fun == BpUnitSpec_cuda::SOFTMAX) { // softmax always uses this!
          u->dEdA += err;
        }
        else { // CROSS_ENTROPY
          if(us->act_fun == BpUnitSpec_cuda::TANH) {
            err /= (u->act + 1.0f) * (1.0f - u->act) * 2.0f;
          }
          else {
            err /= u->act * (1.0f - u->act);
          }
          u->dEdA += err;
        }
      }
    }
    
    u->dEdNet = u->dEdA * us->ActDeriv(u->act);
  }
}

void Bp_cuda::Compute_dEdA_dEdNet() {
  // copy unit spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
    
  const int nlay = n_layers_built;
  for(int li = nlay-1; li >= 0; li--) { // go in reverse order!
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    if(nu > 0) {
      BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
        (units_mem_h, unit_vars_size, st_ui);
      if(u->ext_flag & UnitState_cuda::EXT)
        continue;                 // skip it!  todo: could have bp to inputs but..

      const int n_cgps = units_n_send_cgps_h[st_ui]; // assume same for all..
      if(n_cgps > 0) {
        for(int cgi = 0; cgi < n_cgps; cgi++) {
          // Invoke kernel -- 3rd arg is size of memory to allocate to shared
          // use "act" stream to keep deda and dednet synchronized..
          Kernel_Compute_dEdA_ConState<<<nu, n_threads, n_threads * sizeof(float), strm_compute_act>>>
            (st_ui, cgi, send_cgp_start_d, send_cgp_mem_d, con_group_size,
             send_cons_mem_d, units_mem_d, unit_vars_size,
             recv_cgp_start_d, recv_cgp_mem_d, recv_cons_mem_d);
        }
        cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
      }

      // b/c units are accessed each per a diff thread, the n blocks is divided by threads
      const int n_blocks = (int)ceil((float)nu / (float)n_threads);

      Kernel_Compute_Err_dEdNet<<<n_blocks, n_threads, 0, strm_compute_act>>>
        (st_ui, ed_ui, units_mem_d, unit_vars_size, unit_spec_size,
         send_cgp_start_d, units_n_send_cgps_d, send_cgp_mem_d, con_group_size);
      cudaSafeCall(hipStreamSynchronize(strm_compute_act));
    }
  }
}



//////////////////////////////////////////////////////////////////////////
//      Compute_dWt -- straight by congroups


__global__ void Kernel_Compute_dWt_Bp
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem,
 char* units_mem, const int unit_vars_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpUnitState_cuda* ru = (BpUnitState_cuda*)cg->OwnUnVars(units_mem, unit_vars_size);

  const float ru_dEdNet = ru->dEdNet;
  
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* dwts = cg->OwnCnVar(recv_cons_mem, ConState_cuda::DWT);

  while(st < ed) {
    UnitState_cuda* su = cg->UnVars(recv_cons_mem, units_mem, unit_vars_size, st);
    dwts[st] += su->act * ru_dEdNet;
    st++;
  }
}

__global__ void Kernel_Compute_dWt_Bp_Bias
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size)
{

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);
    u->bias_dwt += u->dEdNet;
  }
}

void Bp_cuda::Compute_dWt(bool sync) {
  // just throw the whole set of recv_cgps at it!

  int n_units_blocks = (int)ceil((float)n_units_built / (float)n_threads);
  
  Kernel_Compute_dWt_Bp<<<n_recv_cgps, n_threads, 0, strm_compute_dwt>>>
    (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, units_mem_d, unit_vars_size);

  Kernel_Compute_dWt_Bp_Bias<<<n_units_blocks, n_threads, 0, strm_compute_dwt_bias>>>
    (0, n_units_built, units_mem_d, unit_vars_size);

  if(sync) {  
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt_bias));
  }
}


//////////////////////////////////////////////////////////////////////////
//      Compute_Weights -- straight by congroups


__global__ void Kernel_Compute_Weights_Bp_dWtOnly
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);

  while(st < ed) {
    wts[st] += cs->cur_lrate * dwts[st];
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_dWtOnly
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      u->bias_wt += cs->cur_lrate * u->bias_dwt;
      u->bias_dwt = 0.0f;
    }
  }
}

__global__ void Kernel_Compute_Weights_Bp_SimpleDecay
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);

  while(st < ed) {
    wts[st] += cs->cur_lrate * (dwts[st] - cs->decay * wts[st]);
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_SimpleDecay
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      u->bias_wt += cs->cur_lrate * (u->bias_dwt - cs->decay * u->bias_wt);
      u->bias_dwt = 0.0f;
    }
  }
}

__global__ void Kernel_Compute_Weights_Bp_Elimination
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);

  while(st < ed) {
    const float wt = wts[st];
    float denom = (1.0f + wt * wt);
    wts[st] += cs->cur_lrate * (dwts[st] - ((cs->decay * wt) / (denom * denom)));
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_Elimination
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      const float wt = u->bias_wt;
      float denom = (1.0f + wt * wt);
      u->bias_wt += cs->cur_lrate * (u->bias_dwt - ((cs->decay * wt) / (denom * denom)));
      u->bias_dwt = 0.0f;
    }
  }
}

__global__ void Kernel_Compute_Weights_Bp_Moment
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);
  float* pdws = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::PDW);

  while(st < ed) {
    pdws[st] = cs->cur_lrate * dwts[st] + cs->momentum * pdws[st];
    wts[st] += pdws[st];
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_Moment
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);

      u->bias_pdw = cs->cur_lrate * u->bias_dwt + cs->momentum * u->bias_pdw;
      u->bias_wt += u->bias_pdw;
      u->bias_dwt = 0.0f;
    }
  }
}

__global__ void Kernel_Compute_Weights_Bp_Moment_Simple
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);
  float* pdws = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::PDW);

  while(st < ed) {
    pdws[st] = cs->cur_lrate * (dwts[st] - cs->decay * wts[st]) + cs->momentum * pdws[st];
    wts[st] += pdws[st];
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_Moment_Simple
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      u->bias_pdw = cs->cur_lrate * (u->bias_dwt - cs->decay * u->bias_wt) +
        cs->momentum * u->bias_pdw;
      u->bias_wt += u->bias_pdw;
      u->bias_dwt = 0.0f;
    }
  }
}

__global__ void Kernel_Compute_Weights_Bp_Moment_Elim
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConState_cuda* cg = Network_cuda::GetConState_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::DWT);
  float* pdws = cg->OwnCnVar(recv_cons_mem, BpConSpec_cuda::PDW);

  while(st < ed) {
    const float wt = wts[st];
    float denom = (1.0f + wt * wt);
    pdws[st] += cs->cur_lrate * (dwts[st] - ((cs->decay * wt) / (denom * denom))) +
      cs->momentum * pdws[st];
    wts[st] += pdws[st];
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_Moment_Elim
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitState_cuda* u = (BpUnitState_cuda*)Network_cuda::GetUnitState
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      const float wt = u->bias_wt;
      float denom = (1.0f + wt * wt);
      u->bias_pdw = cs->cur_lrate * (u->bias_dwt - ((cs->decay * wt) / (denom * denom))) +
        cs->momentum * u->bias_pdw;
      u->bias_wt += u->bias_pdw;
      u->bias_dwt = 0.0f;
    }
  }
}

void Bp_cuda::Compute_Weights(bool sync) {
  // copy con spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), con_spec_mem_d, con_spec_mem_tot);
  // copy unit spec mem to constant, position AFTER con specs (4th arg)
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot, con_spec_mem_tot);
  
  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (con_spec_mem_h, con_spec_size, 0);

  int n_units_blocks = (int)ceil((float)n_units_built / (float)n_threads);
  
  switch(cs->wt_updt) {
  case BpConSpec_cuda::WU_DWT_ONLY: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_dWtOnly<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_SIMPLE_DECAY: {
    Kernel_Compute_Weights_Bp_SimpleDecay<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_SimpleDecay<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_ELIMINATION: {
    Kernel_Compute_Weights_Bp_Elimination<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_Elimination<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT: {
    Kernel_Compute_Weights_Bp_Moment<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_Moment<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT_SIMPLE: {
    Kernel_Compute_Weights_Bp_Moment_Simple<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_Moment_Simple<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT_ELIM: {
    Kernel_Compute_Weights_Bp_Moment_Elim<<<n_recv_cgps, n_threads, 0, strm_compute_weights>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_Moment_Elim<<<n_units_blocks, n_threads, 0, strm_compute_weights_bias>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  }

  if(sync) {                    // generally doesn't have to be sync..
    cudaSafeCall(hipStreamSynchronize(strm_compute_weights));
    cudaSafeCall(hipStreamSynchronize(strm_compute_weights_bias));
  }
}
