#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "bp_cuda.h"

Bp_cuda::Bp_cuda() {
  unit_spec_size = sizeof(BpUnitSpec_cuda);
  con_spec_size = sizeof(BpConSpec_cuda);
}

// this constant memory is used for holding the spec info:

__constant__ char const_spec_mem[Network_cuda::max_constant_mem];


///////////////////////////////////////////////////////////////////////////
//      layer-at-a-time netinput and act

__global__ void Kernel_Compute_Netin_ConGroup
(const int st_ui, const int cgp_idx, int* recv_cgp_start, char* recv_cgp_mem,
 const int con_group_size, float* recv_cons_mem, char* units_mem, const int unit_vars_size) {

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConGroup_cuda* cg = Network_cuda::GetUnConGroup
    (recv_cgp_mem, recv_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  const float* wts = cg->OwnCnVar(recv_cons_mem, ConGroup_cuda::WT);

  float sum = 0.0f;

  while(st < ed) {
    UnitVars_cuda* su = cg->UnVars(recv_cons_mem, units_mem, unit_vars_size, st);
    sum += wts[st] * su->act;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  if(thr_no == 0) {
    sum = 0.0f;
    for(int i=0; i<nthrs; i++) {
      sum += temp_sums[i];
    }
    cg->temp1 = sum;
  }
  //   int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  //   while( i!=0 ) {
  //     if(thr_no < i) {
  //       temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
  //     }
  //     __syncthreads();
  //     i /= 2;                     // binary tree -- only earlier and earlier threads get it
  //   }

  //   if(thr_no == 0) {
  //     cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  //   }
}


__global__ void Kernel_Compute_Act_Bp
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 int unit_spec_size, int* recv_cgp_start, int* units_n_recv_cgps, char* recv_cgp_mem,
 const int con_group_size)
{

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {

    UnitVars_cuda* u = Network_cuda::GetUnitVars(units_mem, unit_vars_size, un_idx);

    if(!(u->ext_flag & UnitVars_cuda::EXT)) {
      // first step is to sum up the netins for this guy
      float netin = 0.0f;
      const int n_cgps = units_n_recv_cgps[un_idx];
      for(int i=0; i<n_cgps; i++) {
        ConGroup_cuda* cg = Network_cuda::GetUnConGroup
          (recv_cgp_mem, recv_cgp_start, con_group_size, un_idx, i);
        netin += cg->temp1;
      }

      netin += u->bias_wt;      // assume bias specs..
      
      BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
        (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);

      u->net = netin;
      u->act = us->ActFromNetin(netin);
      // todo: dropout.. noise..
    }
    else {
      u->act = u->ext;
    }
  }
}

void Bp_cuda::Compute_NetinAct() {
  ExtInputToDevice(true);       // external input comes from host..
      
  // copy unit spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
  
  const int nlay = n_layers_built;
  for(int li=0; li < nlay; li++) {
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    if(nu > 0) {

      const int n_cgps = units_n_recv_cgps_h[st_ui]; // assume same for all..
      if(n_cgps > 0) {
        for(int cgi = 0; cgi < n_cgps; cgi++) {
          //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
          Kernel_Compute_Netin_ConGroup<<<nu, n_threads, n_threads * sizeof(float), strm_compute_netin>>>
            (st_ui, cgi, recv_cgp_start_d, recv_cgp_mem_d, con_group_size,
             recv_cons_mem_d, units_mem_d, unit_vars_size);
        }
        cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
      }

      // b/c units are accessed each per a diff thread, the n blocks is divided by threads
      const int n_blocks = (int)ceil((float)nu / (float)n_threads);

      Kernel_Compute_Act_Bp<<<n_blocks, n_threads, 0, strm_compute_act>>>
        (st_ui, ed_ui, units_mem_d, unit_vars_size, unit_spec_size,
         recv_cgp_start_d, units_n_recv_cgps_d, recv_cgp_mem_d, con_group_size);
      cudaSafeCall(hipStreamSynchronize(strm_compute_act));
    }
  }
}


//////////////////////////////////////////////////////////////////////////
//      layer-at-a-time dEdA and dEdNet


__global__ void Kernel_Compute_dEdA_ConGroup
(const int st_ui, const int cgp_idx, int* send_cgp_start, char* send_cgp_mem,
 const int con_group_size, float* send_cons_mem, char* units_mem, const int unit_vars_size,
 int* recv_cgp_start, char* recv_cgp_mem, float* recv_cons_mem)
{

  extern __shared__ float temp_sums[]; // third arg to kernel specifies size of this!

  const int un_idx = st_ui + blockIdx.x; // blocks are *units*

  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConGroup_cuda* cg = Network_cuda::GetUnConGroup
    (send_cgp_mem, send_cgp_start, con_group_size, un_idx, cgp_idx);

  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float sum = 0.0f;

  while(st < ed) {
    BpUnitVars_cuda* ru = (BpUnitVars_cuda*)cg->UnVars
      (send_cons_mem, units_mem, unit_vars_size, st);
    // this is super deadly slow:
    const float wt = cg->PtrCn
      (send_cons_mem, con_group_size, recv_cgp_start, recv_cgp_mem, recv_cons_mem, st,
       ConGroup_cuda::WT);
    sum += wt * ru->dEdNet;
    st++;
  }

  temp_sums[thr_no] = sum;

  __syncthreads();            // make sure all threads have written to temp_sums

  if(thr_no == 0) {
    sum = 0;
    for(int i=0; i<nthrs; i++) {
      sum += temp_sums[i];
    }
    cg->temp1 = sum;
  }
  // int i = nthrs / 2;            // now use a binary tree aggregation of temp_sums
  // while( i!=0 ) {
  //   if(thr_no < i) {
  //     temp_sums[thr_no] += temp_sums[thr_no + i]; // get from next up
  //   }
  //   __syncthreads();
  //   i /= 2;                     // binary tree -- only earlier and earlier threads get it
  // }

  // if(thr_no == 0) {
  //   cg->temp1 = temp_sums[0]; // first guy has it all, store into our con group for later summation
  // }
}

__global__ void Kernel_Compute_Err_dEdNet
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size, int unit_spec_size,
 int* send_cgp_start, int* units_n_send_cgps, char* send_cgp_mem, const int con_group_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;

  if(un_idx < ed_ui) {
    
    BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem, unit_vars_size, un_idx);

    // first step is to sum up the dEdA's for this guy
    float dEdA = 0.0f;
    const int n_cgps = units_n_send_cgps[un_idx];
    for(int i=0; i<n_cgps; i++) {
      ConGroup_cuda* cg = Network_cuda::GetUnConGroup
        (send_cgp_mem, send_cgp_start, con_group_size, un_idx, i);
      dEdA += cg->temp1;
    }

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem, unit_spec_size, u->cuda_unit_spec_idx);
    
    u->dEdA = dEdA;

    // compute error
    if(u->ext_flag & UnitVars_cuda::TARG) {
      float err = u->targ - u->act;
      if(fabsf(err) >= us->err_tol) {
        if(us->error_fun == BpUnitSpec_cuda::SQUARED_ERR
           || us->act_fun == BpUnitSpec_cuda::SOFTMAX) { // softmax always uses this!
          u->dEdA += err;
        }
        else { // CROSS_ENTROPY
          if(us->act_fun == BpUnitSpec_cuda::TANH) {
            err /= (u->act + 1.0f) * (1.0f - u->act) * 2.0f;
          }
          else {
            err /= u->act * (1.0f - u->act);
          }
          u->dEdA += err;
        }
      }
    }
    
    u->dEdNet = u->dEdA * us->ActDeriv(u->act);
  }
}

void Bp_cuda::Compute_dEdA_dEdNet() {
  // copy unit spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot);
    
  const int nlay = n_layers_built;
  for(int li = nlay-1; li >= 0; li--) { // go in reverse order!
    const int st_ui = LayUnStart(lay_unit_idxs_h, li);
    const int ed_ui = LayUnEnd(lay_unit_idxs_h, li);
    const int nu = ed_ui - st_ui;

    if(nu > 0) {
      BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
        (units_mem_h, unit_vars_size, st_ui);
      if(u->ext_flag & UnitVars_cuda::EXT)
        continue;                 // skip it!  todo: could have bp to inputs but..

      const int n_cgps = units_n_send_cgps_h[st_ui]; // assume same for all..
      if(n_cgps > 0) {
        for(int cgi = 0; cgi < n_cgps; cgi++) {
          //  Invoke kernel -- 3rd arg is size of memory to allocate to shared
          Kernel_Compute_dEdA_ConGroup<<<nu, n_threads, n_threads * sizeof(float), strm_compute_netin>>>
            (st_ui, cgi, send_cgp_start_d, send_cgp_mem_d, con_group_size,
             send_cons_mem_d, units_mem_d, unit_vars_size,
             recv_cgp_start_d, recv_cgp_mem_d, recv_cons_mem_d);
        }
        cudaSafeCall(hipStreamSynchronize(strm_compute_netin));
      }

      // b/c units are accessed each per a diff thread, the n blocks is divided by threads
      const int n_blocks = (int)ceil((float)nu / (float)n_threads);

      Kernel_Compute_Err_dEdNet<<<n_blocks, n_threads, 0, strm_compute_act>>>
        (st_ui, ed_ui, units_mem_d, unit_vars_size, unit_spec_size,
         send_cgp_start_d, units_n_send_cgps_d, send_cgp_mem_d, con_group_size);
      cudaSafeCall(hipStreamSynchronize(strm_compute_act));
    }
  }

  TargUnitsToHost(true);       // send output layer data back to host for stats..
}



//////////////////////////////////////////////////////////////////////////
//      Compute_dWt -- straight by congroups


__global__ void Kernel_Compute_dWt_Bp
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem,
 char* units_mem, const int unit_vars_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpUnitVars_cuda* ru = (BpUnitVars_cuda*)cg->OwnUnVars(units_mem, unit_vars_size);

  const float ru_dEdNet = ru->dEdNet;
  
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* dwts = cg->OwnCnVar(recv_cons_mem, ConGroup_cuda::DWT);

  while(st < ed) {
    UnitVars_cuda* su = cg->UnVars(recv_cons_mem, units_mem, unit_vars_size, st);
    dwts[st] += su->act * ru_dEdNet;
    st++;
  }
}

__global__ void Kernel_Compute_dWt_Bp_Bias
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size)
{

  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem, unit_vars_size, un_idx);
    u->bias_dwt = u->dEdNet;
  }
}

void Bp_cuda::Compute_dWt(bool sync) {
  // just throw the whole set of recv_cgps at it!
  Kernel_Compute_dWt_Bp<<<n_recv_cgps, n_threads, 0, strm_compute_dwt>>>
    (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, units_mem_d, unit_vars_size);

  Kernel_Compute_dWt_Bp_Bias<<<n_units_built, n_threads, 0, strm_compute_dwt>>>
    (0, n_units_built, units_mem_d, unit_vars_size);

  if(sync) {  
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
  }
}


//////////////////////////////////////////////////////////////////////////
//      Compute_Weights -- straight by congroups


__global__ void Kernel_Compute_Weights_Bp_dWtOnly
(char* recv_cgp_mem, const int con_group_size, float* recv_cons_mem, int con_spec_size) {

  const int cgp_idx = blockIdx.x; // blocks are connection groups
  const int nthrs = blockDim.x; // threads are connections
  const int thr_no = threadIdx.x;
    
  ConGroup_cuda* cg = Network_cuda::GetConGroup_Flat(recv_cgp_mem, con_group_size, cgp_idx);

  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (const_spec_mem, con_spec_size, cg->con_spec_idx);
    
  const int sz = cg->size;
  int st, ed;
  Network_cuda::GetThreadCons(nthrs, thr_no, sz, st, ed);
     
  float* wts = cg->OwnCnVar(recv_cons_mem, ConGroup_cuda::WT);
  float* dwts = cg->OwnCnVar(recv_cons_mem, ConGroup_cuda::DWT);

  while(st < ed) {
    wts[st] += cs->cur_lrate * dwts[st];
    dwts[st] = 0.0f;
    st++;
  }
}

__global__ void Kernel_Compute_Weights_Bp_Bias_dWtOnly
(const int st_ui, const int ed_ui, char* units_mem, const int unit_vars_size,
 const int unit_spec_size, const int con_spec_mem_tot, const int con_spec_size)
{
  // each thread just gets a different unit -- doesn't do multiple units
  const int nthrs = blockDim.x;
  const int thr_no = threadIdx.x;
  const int un_idx = st_ui + blockIdx.x * nthrs + thr_no;
  if(un_idx < ed_ui) {
    BpUnitVars_cuda* u = (BpUnitVars_cuda*)Network_cuda::GetUnitVars
      (units_mem, unit_vars_size, un_idx);

    BpUnitSpec_cuda* us = (BpUnitSpec_cuda*)Network_cuda::GetUnitSpec
      (const_spec_mem + con_spec_mem_tot, unit_spec_size, u->cuda_unit_spec_idx);

    if(us->bias_spec_idx >= 0) {
      BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
        (const_spec_mem, con_spec_size, us->bias_spec_idx);
      
      u->bias_wt += cs->cur_lrate * u->bias_dwt;
    }
  }
}

void Bp_cuda::Compute_Weights(bool sync) {
  // copy con spec mem to constant
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), con_spec_mem_d, con_spec_mem_tot);

  // copy unit spec mem to constant, position AFTER con specs (4th arg)
  hipMemcpyToSymbol(HIP_SYMBOL(const_spec_mem), unit_spec_mem_d, unit_spec_mem_tot, con_spec_mem_tot);
  
  BpConSpec_cuda* cs = (BpConSpec_cuda*)Network_cuda::GetConSpec
    (con_spec_mem_h, con_spec_size, 0);

  switch(cs->wt_updt) {
  case BpConSpec_cuda::WU_DWT_ONLY: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    Kernel_Compute_Weights_Bp_Bias_dWtOnly<<<n_units_built, n_threads, 0, strm_compute_wt>>>
      (0, n_units_built, units_mem_d, unit_vars_size, unit_spec_size, con_spec_mem_tot,
       con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_SIMPLE_DECAY: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_ELIMINATION: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT_SIMPLE: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    break;
  }
  case BpConSpec_cuda::WU_MOMENT_ELIM: {
    Kernel_Compute_Weights_Bp_dWtOnly<<<n_recv_cgps, n_threads, 0, strm_compute_wt>>>
      (recv_cgp_mem_d, con_group_size, recv_cons_mem_d, con_spec_size);
    break;
  }
  }

  if(sync) {                    // generally doesn't have to be sync..
    cudaSafeCall(hipStreamSynchronize(strm_compute_wt));
  }
}
