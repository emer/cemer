
#include <hip/hip_runtime.h>
// cuda file

__global__ void VecAdd(float* A, float* B, float* C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

void DoAdd() {
  int N = 256;
  size_t size = N * sizeof(float);

  // Allocate input vectors h_A and h_B in host memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);
  // Initialize input vectors ...
  
  // Allocate vectors in device memory
  float* d_A;
  hipMalloc(&d_A, size);
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);
  
  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);
}
