#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "LeabraConSpec_cuda.h"

LeabraConSpecCuda::LeabraConSpecCuda() {
  Initialize();
}

LeabraConSpecCuda::~LeabraConSpecCuda() {
  FreeCudaArrays();
}

void LeabraConSpecCuda::Initialize() {
  n_units = 0;
  own_cons_max_size = 0;
  thread_chunk_sz = 8;
  max_threads = 0;
  n_threads = 256;
  own_cons_cnt = 0;
  ptr_cons_cnt = 0;
  own_units_x_cons = 0;
  ptr_units_x_cons = 0;

  own_cons_mem_h = NULL;
  own_cons_mem_d = NULL;
  ptr_cons_mem_h = NULL;
  ptr_cons_mem_d = NULL;

  units_h = NULL;
  units_d = NULL;
  con_mem_idxs_h = NULL;
  con_mem_idxs_d = NULL;
  con_allocs_h = NULL;
  con_allocs_d = NULL;
  con_sizes_h = NULL;
  con_sizes_d = NULL;
  unit_starts_h = NULL;

  cur_send_net_n = 0;
  cur_send_net_h = NULL;
  cur_send_net_d = NULL;
  send_net_acts_h = NULL;
  send_net_acts_d = NULL;
  send_netin_tmp_h = NULL;
  send_netin_tmp_d = NULL;
}

void LeabraConSpecCuda::FreeCudaArrays() {
  if(own_cons_mem_d)
    hipFree(own_cons_mem_d);
  if(ptr_cons_mem_d)
    hipFree(ptr_cons_mem_d);

  if(units_h)
    free(units_h);
  if(units_d)
    hipFree(units_d);

  if(con_mem_idxs_h)
    free(con_mem_idxs_h);
  if(con_mem_idxs_d)
    hipFree(con_mem_idxs_d);

  if(con_allocs_h)
    free(con_allocs_h);
  if(con_allocs_d)
    hipFree(con_allocs_d);

  if(con_sizes_h)
    free(con_sizes_h);
  if(con_sizes_d)
    hipFree(con_sizes_d);

  if(unit_starts_h)
    free(unit_starts_h);

  if(cur_send_net_h)
    free(cur_send_net_h);
  if(cur_send_net_d)
    hipFree(cur_send_net_d);

  if(send_net_acts_h)
    free(send_net_acts_h);
  if(send_net_acts_d)
    hipFree(send_net_acts_d);

  if(send_netin_tmp_d)
    hipFree(send_netin_tmp_d);

  Initialize();
}

void LeabraConSpecCuda::AllocCudaArrays
(int n_un, int own_cons_max_sz, int64_t own_cnt, int64_t ptr_cnt,
 int own_units_x, int ptr_units_x, 
 float* own_cons_mem, float* ptr_cons_mem, float* send_netin_tmp) {
  if(n_units != n_un || own_units_x != own_units_x_cons) {
    FreeCudaArrays();
  }

  if(n_un == 0 || own_units_x == 0)
    return;

  n_units = n_un;
  own_cons_max_size = own_cons_max_sz;
  thread_chunk_sz = 8;
  max_threads = own_cons_max_size / thread_chunk_sz;

  // docs on number of threads: http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#execution-configuration-optimizations

  int mod32 = max_threads % 32;
  if(mod32 != 0)
    n_threads = ((max_threads / 32) + 1) * 32;
  else
    n_threads = max_threads;
  if(n_threads > 256)
    n_threads = 256;

  own_cons_cnt = own_cnt;
  ptr_cons_cnt = ptr_cnt;
  own_units_x_cons = own_units_x;
  ptr_units_x_cons = ptr_units_x;

  own_cons_mem_h = own_cons_mem;
  ptr_cons_mem_h = ptr_cons_mem;
  send_netin_tmp_h = send_netin_tmp;

  units_h = (int*)malloc(own_units_x_cons * sizeof(int));
  hipMalloc(&units_d, own_units_x_cons * sizeof(int));

  con_mem_idxs_h = (int64_t*)malloc(own_units_x_cons * sizeof(int64_t));
  hipMalloc(&con_mem_idxs_d, own_units_x_cons * sizeof(int64_t));

  con_allocs_h = (int*)malloc(own_units_x_cons * sizeof(int));
  hipMalloc(&con_allocs_d, own_units_x_cons * sizeof(int));

  con_sizes_h = (int*)malloc(own_units_x_cons * sizeof(int));
  hipMalloc(&con_sizes_d, own_units_x_cons * sizeof(int));

  unit_starts_h = (int*)malloc((n_units+1) * sizeof(int));

  cur_send_net_h = (int*)malloc(own_units_x_cons * sizeof(int));
  hipMalloc(&cur_send_net_d, own_units_x_cons * sizeof(int));

  send_net_acts_h = (float*)malloc(own_units_x_cons * sizeof(float));
  hipMalloc(&send_net_acts_d, own_units_x_cons * sizeof(float));

  hipMalloc(&send_netin_tmp_d, (n_units+1) * sizeof(float));

  hipMalloc(&own_cons_mem_d, own_cons_cnt * sizeof(float));

  // conserve memory: not needed..
  //   hipMalloc(&ptr_cons_mem_d, ptr_cons_cnt);
}

void LeabraConSpecCuda::UpdateOwnCons() {
  if(own_cons_mem_h && own_cons_mem_d) {
    hipMemcpy(own_cons_mem_d, own_cons_mem_h, own_cons_cnt * sizeof(float),
               hipMemcpyHostToDevice);
  }
}

void LeabraConSpecCuda::UpdateUnitsXCons() {
  if(!units_h) return;

  float sz =  own_units_x_cons * sizeof(float);

  hipMemcpy(units_d, units_h, sz, hipMemcpyHostToDevice);
  hipMemcpy(con_mem_idxs_d, con_mem_idxs_h, sz, hipMemcpyHostToDevice);
  hipMemcpy(con_allocs_d, con_allocs_h, sz, hipMemcpyHostToDevice);
  hipMemcpy(con_sizes_d, con_sizes_h, sz, hipMemcpyHostToDevice);
}

__global__ void Kernel_Send_NetinDelta
(int cur_send_net_n, int* cur_send_net_d, float* send_net_acts_d, float* send_netin_tmp_d,
 float* own_cons_mem_d, int64_t* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d) {
  int csni = blockIdx.x;
  int nth = blockDim.x;
  if (csni < cur_send_net_n) {
    int ucidx = cur_send_net_d[csni];
    float send_eff = send_net_acts_d[csni];
    const int sz = con_sizes_d[ucidx];
    const float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
      (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::WT));
    const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
    int th = threadIdx.x;
    int cn_per_th = (sz / nth) + 1; // round up
    int st = th * cn_per_th;
    int ed = st + cn_per_th;
    ed = ed < sz ? ed : sz;     // max of sz
    while(st < ed) {
      int ridx = ridxs[st];
      atomicAdd(&(send_netin_tmp_d[ridx]), wts[st] * send_eff);
      st++;
    }
  }
}

void Test_Kernel_Send_NetinDelta
(int blockIdx, int blockDim, int threadIdx,
 int cur_send_net_n, int* cur_send_net_d, float* send_net_acts_d, float* send_netin_tmp_d,
 float* own_cons_mem_d, int64_t* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d) {
  int csni = blockIdx;
  int nth = blockDim;
  if (csni < cur_send_net_n) {
    int ucidx = cur_send_net_d[csni];
    float send_eff = send_net_acts_d[csni];
    const int sz = con_sizes_d[ucidx];
    const float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
      (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::WT));
    const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
    int th = threadIdx;
    int cn_per_th = (sz / nth) + 1; // round up
    int st = th * cn_per_th;
    int ed = st + cn_per_th;
    ed = ed < sz ? ed : sz;     // max of sz
    while(st < ed) {
      int ridx = ridxs[st];
      send_netin_tmp_d[ridx] += wts[st] * send_eff;
      st++;
    }
  }
}


void LeabraConSpecCuda::Send_NetinDelta() {
  if(cur_send_net_n == 0) return;

  float sz = cur_send_net_n * sizeof(float);

  hipMemcpy(cur_send_net_d, cur_send_net_h, sz, hipMemcpyHostToDevice);
  hipMemcpy(send_net_acts_d, send_net_acts_h, sz, hipMemcpyHostToDevice);

  hipMemset(send_netin_tmp_d, 0, (n_units+1) * sizeof(float));

  // manual testing for getting all the logic right
  // for(int i=0; i< cur_send_net_n; i++) {
  //   for(int j=0; j< n_threads; j++) {
  //     Test_Kernel_Send_NetinDelta
  //       (i, n_threads, j, cur_send_net_n, cur_send_net_h, send_net_acts_h,
  //        send_netin_tmp_h, own_cons_mem_h, con_mem_idxs_h, con_allocs_h, con_sizes_h);
  //   }
  // }

  //  Invoke kernel
  Kernel_Send_NetinDelta<<<cur_send_net_n, n_threads>>>
    (cur_send_net_n, cur_send_net_d, send_net_acts_d, send_netin_tmp_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d);

  hipDeviceSynchronize();

  hipMemcpy(send_netin_tmp_h, send_netin_tmp_d, (n_units+1) * sizeof(float),
             hipMemcpyDeviceToHost);
  // get results back from device -- args are reversed here!
}
