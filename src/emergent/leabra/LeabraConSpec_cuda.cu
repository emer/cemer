#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "LeabraConSpec_cuda.h"

//  lldb print commands -- use memory read with format
//  me r -rf f -l148 -c148 send_netin_tmp_h

LeabraConSpecCuda::LeabraConSpecCuda() {
  Initialize();
}

LeabraConSpecCuda::~LeabraConSpecCuda() {
  FreeCudaArrays();
}

void LeabraConSpecCuda::Initialize() {
  n_units = 0;
  own_cons_max_size = 0;
  thread_chunk_sz = 8;
  max_threads = 0;
  n_threads = 256;
  own_cons_cnt = 0;
  ptr_cons_cnt = 0;
  own_units_x_cons = 0;
  ptr_units_x_cons = 0;

  strms_created = false;

  own_cons_mem_h = NULL;
  own_cons_mem_d = NULL;
  ptr_cons_mem_h = NULL;
  ptr_cons_mem_d = NULL;

  units_h = NULL;
  units_d = NULL;
  con_mem_idxs_h = NULL;
  con_mem_idxs_d = NULL;
  con_allocs_h = NULL;
  con_allocs_d = NULL;
  con_sizes_h = NULL;
  con_sizes_d = NULL;
  con_recv_idxs_h = NULL;
  con_recv_idxs_d = NULL;
  unit_starts_h = NULL;

  con_params_h = NULL;
  con_params_d = NULL;
  wt_sig_fun_h = NULL;
  wt_sig_fun_d = NULL;

  cur_units_x_cons_n = 0;
  cur_units_x_cons_h = NULL;
  cur_units_x_cons_d = NULL;
  send_net_acts_h = NULL;
  send_net_acts_d = NULL;
  send_net_max_prjns = 1;
  send_netin_tmp_h = NULL;
  send_netin_tmp_d = NULL;
  send_d5bnet_tmp_h = NULL;
  send_d5bnet_tmp_d = NULL;

  unit_vec_vars_h = NULL;
  unit_vec_vars_d = NULL;
}

void LeabraConSpecCuda::FreeCudaArrays() {
  if(own_cons_mem_d)
    hipFree(own_cons_mem_d);
  if(ptr_cons_mem_d)
    hipFree(ptr_cons_mem_d);

  if(units_h)
    free(units_h);
  if(units_d)
    hipFree(units_d);

  if(con_mem_idxs_h)
    free(con_mem_idxs_h);
  if(con_mem_idxs_d)
    hipFree(con_mem_idxs_d);

  if(con_allocs_h)
    free(con_allocs_h);
  if(con_allocs_d)
    hipFree(con_allocs_d);

  if(con_sizes_h)
    free(con_sizes_h);
  if(con_sizes_d)
    hipFree(con_sizes_d);

  if(con_recv_idxs_h)
    free(con_recv_idxs_h);
  if(con_recv_idxs_d)
    hipFree(con_recv_idxs_d);

  if(unit_starts_h)
    free(unit_starts_h);

  if(con_params_h)
    free(con_params_h);
  if(con_params_d)
    hipFree(con_params_d);

  if(wt_sig_fun_d)
    hipFree(wt_sig_fun_d);

  if(cur_units_x_cons_h)
    free(cur_units_x_cons_h);
  if(cur_units_x_cons_d)
    hipFree(cur_units_x_cons_d);

  if(send_net_acts_h)
    free(send_net_acts_h);
  if(send_net_acts_d)
    hipFree(send_net_acts_d);

  if(send_netin_tmp_d)
    hipFree(send_netin_tmp_d);

  if(send_d5bnet_tmp_d)
    hipFree(send_d5bnet_tmp_d);

  if(unit_vec_vars_d)
    hipFree(unit_vec_vars_d);

  if(strms_created) {
    hipStreamDestroy(strm_updt_cons);
    hipStreamDestroy(strm_send_netin);
    hipStreamDestroy(strm_compute_dwt);
    hipStreamDestroy(strm_compute_wt);
    strms_created = false;
  }

  Initialize();
}

void LeabraConSpecCuda::AllocCudaArrays
(int n_un, int own_cons_max_sz, bigint own_cnt, bigint ptr_cnt,
 int own_units_x, int ptr_units_x, 
 float* own_cons_mem, float* ptr_cons_mem, float* send_netin_tmp, int send_net_max_prj,
 float* send_d5bnet_tmp, float* unit_vec_vars)
{
  if(n_un == n_units && own_units_x == own_units_x_cons && own_cnt == own_cons_cnt)
    return;                     // already allocated

  FreeCudaArrays();

  if(n_un == 0 || own_units_x == 0)
    return;

  hipStreamCreate(&strm_updt_cons);
  hipStreamCreate(&strm_send_netin);
  hipStreamCreate(&strm_compute_dwt);
  hipStreamCreate(&strm_compute_wt);
  strms_created = true;

  n_units = n_un;
  own_cons_max_size = own_cons_max_sz;
  thread_chunk_sz = 8;
  max_threads = own_cons_max_size / thread_chunk_sz;

  // docs on number of threads: http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#execution-configuration-optimizations

  int mod32 = max_threads % 32;
  if(mod32 != 0)
    n_threads = ((max_threads / 32) + 1) * 32;
  else
    n_threads = max_threads;
  if(n_threads > 256)
    n_threads = 256;

  own_cons_cnt = own_cnt;
  ptr_cons_cnt = ptr_cnt;
  own_units_x_cons = own_units_x;
  ptr_units_x_cons = ptr_units_x;

  own_cons_mem_h = own_cons_mem;
  ptr_cons_mem_h = ptr_cons_mem;
  send_netin_tmp_h = send_netin_tmp;
  send_net_max_prjns = send_net_max_prj;
  send_d5bnet_tmp_h = send_d5bnet_tmp;
  unit_vec_vars_h = unit_vec_vars;

  units_h = (int*)malloc(own_units_x_cons * sizeof(int));
  cudaSafeCall(hipMalloc(&units_d, own_units_x_cons * sizeof(int)));

  con_mem_idxs_h = (bigint*)malloc(own_units_x_cons * sizeof(bigint));
  cudaSafeCall(hipMalloc(&con_mem_idxs_d, own_units_x_cons * sizeof(bigint)));

  con_allocs_h = (int*)malloc(own_units_x_cons * sizeof(int));
  cudaSafeCall(hipMalloc(&con_allocs_d, own_units_x_cons * sizeof(int)));

  con_sizes_h = (int*)malloc(own_units_x_cons * sizeof(int));
  cudaSafeCall(hipMalloc(&con_sizes_d, own_units_x_cons * sizeof(int)));

  con_recv_idxs_h = (int*)malloc(own_units_x_cons * sizeof(int));
  cudaSafeCall(hipMalloc(&con_recv_idxs_d, own_units_x_cons * sizeof(int)));

  unit_starts_h = (int*)malloc((n_units+1) * sizeof(int));

  con_params_h = (float*)malloc(own_units_x_cons * N_CON_PARAMS * sizeof(float));
  cudaSafeCall(hipMalloc(&con_params_d, own_units_x_cons * N_CON_PARAMS * sizeof(float)));

  cudaSafeCall(hipMalloc(&wt_sig_fun_d, 10002 * sizeof(float)));

  cur_units_x_cons_h = (int*)malloc(own_units_x_cons * sizeof(int));
  cudaSafeCall(hipMalloc(&cur_units_x_cons_d, own_units_x_cons * sizeof(int)));

  send_net_acts_h = (float*)malloc(own_units_x_cons * sizeof(float));
  cudaSafeCall(hipMalloc(&send_net_acts_d, own_units_x_cons * sizeof(float)));

  cudaSafeCall(hipMalloc(&send_netin_tmp_d,
                          (n_units+1) * send_net_max_prjns * sizeof(float)));

  cudaSafeCall(hipMalloc(&send_d5bnet_tmp_d, (n_units+1) * sizeof(float)));

  cudaSafeCall(hipMalloc(&own_cons_mem_d, own_cons_cnt * sizeof(float)));

  cudaSafeCall(hipMalloc(&unit_vec_vars_d, (n_units+1) * N_VEC_VARS * sizeof(float)));

  // conserve memory: not needed..
  //   hipMalloc(&ptr_cons_mem_d, ptr_cons_cnt);
}

void LeabraConSpecCuda::OwnCons_HostToDevice(bool sync) {
  if(own_cons_mem_h && own_cons_mem_d) {
    cudaSafeCall(hipMemcpyAsync(own_cons_mem_d, own_cons_mem_h,
                                 own_cons_cnt * sizeof(float),
                                 hipMemcpyHostToDevice, strm_updt_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_updt_cons));
    }
  }
}

void LeabraConSpecCuda::OwnCons_DeviceToHost(bool sync) {
  if(own_cons_mem_h && own_cons_mem_d) {
    cudaSafeCall(hipMemcpyAsync(own_cons_mem_h, own_cons_mem_d,
                                 own_cons_cnt * sizeof(float),
                                 hipMemcpyDeviceToHost, strm_updt_cons));
    if(sync) {
      cudaSafeCall(hipStreamSynchronize(strm_updt_cons));
    }
  }
}

void LeabraConSpecCuda::UpdateUnitsXCons() {
  if(!con_mem_idxs_h) return;

  int sz =  own_units_x_cons * sizeof(int);

  cudaSafeCall(hipMemcpy(units_d, units_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_mem_idxs_d, con_mem_idxs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_allocs_d, con_allocs_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_sizes_d, con_sizes_h, sz, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(con_recv_idxs_d, con_recv_idxs_h, sz, hipMemcpyHostToDevice));
}

void LeabraConSpecCuda::UpdateConParams() {
  if(!con_params_h) return;

  int sz =  own_units_x_cons * N_CON_PARAMS * sizeof(float);

  cudaSafeCall(hipMemcpy(con_params_d, con_params_h, sz, hipMemcpyHostToDevice));

  if(wt_sig_fun_h) {
    cudaSafeCall(hipMemcpy(wt_sig_fun_d, wt_sig_fun_h, 10002 * sizeof(float),
                            hipMemcpyHostToDevice));
  }
}

__global__ void Kernel_Send_NetinDelta
(int* cur_units_x_cons_d, float* send_net_acts_d, float* send_netin_tmp_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const float send_eff = send_net_acts_d[csni];
  const int sz = con_sizes_d[ucidx];
  const float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::WT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  const int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    atomicAdd(&(send_netin_tmp_d[ridx]), wts[st] * send_eff);
    // send_netin_tmp_d[ridx] += wts[st] * send_eff; // determine effect of atomic -- not much penalty there
    st++;
  }
}

__global__ void Kernel_Send_NetinDelta_per_prj
(int* cur_units_x_cons_d, float* send_net_acts_d, float* send_netin_tmp_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 int* con_recv_idxs_d, const int nu) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const float send_eff = send_net_acts_d[csni];
  const int sz = con_sizes_d[ucidx];
  const int recv_idx = con_recv_idxs_d[ucidx];
  float* send_netin_vec = send_netin_tmp_d + recv_idx * nu; // nu = n_units+1
  const float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::WT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  const int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    atomicAdd(&(send_netin_vec[ridx]), wts[st] * send_eff);
    st++;
  }
}

void LeabraConSpecCuda::Send_NetinDelta() {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemsetAsync(send_netin_tmp_d, 0,
                               (n_units+1) * send_net_max_prjns * sizeof(float),
                               strm_send_netin));

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_send_netin));
  cudaSafeCall(hipMemcpyAsync(send_net_acts_d, send_net_acts_h,
                               cur_units_x_cons_n * sizeof(float),
                               hipMemcpyHostToDevice, strm_send_netin));

  if(send_net_max_prjns > 1) {
    //  Invoke kernel
    Kernel_Send_NetinDelta_per_prj<<<cur_units_x_cons_n, n_threads, 0, strm_send_netin>>>
      (cur_units_x_cons_d, send_net_acts_d, send_netin_tmp_d,
       own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, con_recv_idxs_d,
       n_units+1);
  }
  else {
    //  Invoke kernel
    Kernel_Send_NetinDelta<<<cur_units_x_cons_n, n_threads, 0, strm_send_netin>>>
      (cur_units_x_cons_d, send_net_acts_d, send_netin_tmp_d,
       own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d);
  }

  cudaSafeCall(hipMemcpyAsync(send_netin_tmp_h, send_netin_tmp_d,
                               (n_units+1) * send_net_max_prjns * sizeof(float),
                               hipMemcpyDeviceToHost, strm_send_netin));
  // get results back from device -- args are reversed here!

  cudaSafeCall(hipStreamSynchronize(strm_send_netin));
}


void LeabraConSpecCuda::Send_Deep5bNetinDelta() {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemsetAsync(send_d5bnet_tmp_d, 0, (n_units+1) * sizeof(float),
                               strm_send_netin));

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_send_netin));
  cudaSafeCall(hipMemcpyAsync(send_net_acts_d, send_net_acts_h,
                               cur_units_x_cons_n * sizeof(float),
                               hipMemcpyHostToDevice, strm_send_netin));

  //  Invoke kernel
  Kernel_Send_NetinDelta<<<cur_units_x_cons_n, n_threads, 0, strm_send_netin>>>
    (cur_units_x_cons_d, send_net_acts_d, send_d5bnet_tmp_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d);

  cudaSafeCall(hipMemcpyAsync(send_d5bnet_tmp_h, send_d5bnet_tmp_d,
                               (n_units+1) * sizeof(float),
                               hipMemcpyDeviceToHost, strm_send_netin));
  // get results back from device -- args are reversed here!

  cudaSafeCall(hipStreamSynchronize(strm_send_netin));
}

void LeabraConSpecCuda::Send_TICtxtNetin() {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemsetAsync(send_netin_tmp_d, 0, (n_units+1) * sizeof(float),
                               strm_send_netin));

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_send_netin));
  cudaSafeCall(hipMemcpyAsync(send_net_acts_d, send_net_acts_h,
                               cur_units_x_cons_n * sizeof(float),
                               hipMemcpyHostToDevice, strm_send_netin));

  //  Invoke kernel
  Kernel_Send_NetinDelta<<<cur_units_x_cons_n, n_threads, 0, strm_send_netin>>>
    (cur_units_x_cons_d, send_net_acts_d, send_netin_tmp_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d);

  cudaSafeCall(hipMemcpyAsync(send_netin_tmp_h, send_netin_tmp_d,
                               (n_units+1) * sizeof(float),
                               hipMemcpyDeviceToHost, strm_send_netin));
  // get results back from device -- args are reversed here!

  cudaSafeCall(hipStreamSynchronize(strm_send_netin));
}


__global__ void Kernel_Compute_dWt_cosdif
(int* cur_units_x_cons_d, float* unit_vec_vars_d, float* con_params_d, int* units_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 const int nu) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const int sidx = units_d[ucidx];

  const float su_avg_s = unit_vec_vars_d[LeabraConSpecCuda::AVG_S * nu + sidx];
  const float su_avg_m = unit_vec_vars_d[LeabraConSpecCuda::AVG_M * nu + sidx];

  const float s_mix = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                   LeabraConSpecCuda::S_MIX];
  const float m_mix = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                   LeabraConSpecCuda::M_MIX];
  const float thr_l_mix = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                       LeabraConSpecCuda::THR_L_MIX];
  const float thr_max = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                     LeabraConSpecCuda::THR_MAX];
  const float clrate = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                    LeabraConSpecCuda::CUR_LRATE];

  const int sz = con_sizes_d[ucidx];
  float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::DWT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    const float ru_avg_s = unit_vec_vars_d[LeabraConSpecCuda::AVG_S * nu + ridx];
    const float ru_avg_m = unit_vec_vars_d[LeabraConSpecCuda::AVG_M * nu + ridx];
    const float ru_avg_l = unit_vec_vars_d[LeabraConSpecCuda::AVG_L * nu + ridx];

    // unfortunately, cos_diff_lmix is on recv layer -- so this needs to be in the loop
    // whereas normally it is outside the loop.. would require a separate
    // var array just for it, at the unit_x_con level..  could look into it later
    const float cos_diff_lmix = unit_vec_vars_d[LeabraConSpecCuda::COS_DIFF_LMIX * nu
                                                + ridx];
    const float efflmix = thr_l_mix * cos_diff_lmix;
    const float effmmix = 1.0f - efflmix;
    const float su_act_mult = efflmix * su_avg_m;

    const float srs = ru_avg_s * su_avg_s;
    const float srm = ru_avg_m * su_avg_m;
    const float sm_mix = s_mix * srs + m_mix * srm;
    const float lthr = su_act_mult * ru_avg_l;
    float effthr = effmmix * srm + lthr;
    effthr = effthr < thr_max ? effthr : thr_max; // max = thr_max

    float rval;                 // xcal.dWtFun
    if(sm_mix < 0.0001f)        // d_thr = 0.0001
      rval = 0.0f;
    else if(sm_mix > effthr * 0.1f) // d_rev = 0.1
      rval = (sm_mix - effthr);
    else
      rval = sm_mix * -9.0f;    // d_rev_ratio = -9.0;
    dwts[st] += clrate * rval;
    st++;
  }
}

void LeabraConSpecCuda::Compute_dWt(bool sync) {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_compute_dwt));
  cudaSafeCall(hipMemcpyAsync(unit_vec_vars_d, unit_vec_vars_h,
                               (n_units+1) * N_VEC_VARS * sizeof(float),
                               hipMemcpyHostToDevice, strm_compute_dwt));

  //  Invoke kernel
  Kernel_Compute_dWt_cosdif<<<cur_units_x_cons_n, n_threads, 0, strm_compute_dwt>>>
    (cur_units_x_cons_d, unit_vec_vars_d, con_params_d, units_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, n_units+1);

  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
  }
}

__global__ void Kernel_Compute_dWt_TICtxt
(int* cur_units_x_cons_d, float* unit_vec_vars_d, float* con_params_d, int* units_d,
 float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 const int nu) {
  const int csni = blockIdx.x;
  const int nth = blockDim.x;
  const int ucidx = cur_units_x_cons_d[csni];
  const int sidx = units_d[ucidx];

  const float su_act_q0 = unit_vec_vars_d[LeabraConSpecCuda::ACT_Q0 * nu + sidx];

  const float clrate = con_params_d[ucidx * LeabraConSpecCuda::N_CON_PARAMS +
                                    LeabraConSpecCuda::CUR_LRATE];

  const int sz = con_sizes_d[ucidx];
  float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::DWT));
  const int* ridxs = ((int*)own_cons_mem_d) + con_mem_idxs_d[ucidx];
  int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    int ridx = ridxs[st];
    const float ru_avg_s = unit_vec_vars_d[LeabraConSpecCuda::AVG_S * nu + ridx];
    const float ru_avg_m = unit_vec_vars_d[LeabraConSpecCuda::AVG_M * nu + ridx];

    dwts[st] += clrate * (ru_avg_s - ru_avg_m) * su_act_q0;
    st++;
  }
}

void LeabraConSpecCuda::Compute_dWt_TICtxt(bool sync) {
  if(cur_units_x_cons_n == 0) return;

  cudaSafeCall(hipMemcpyAsync(cur_units_x_cons_d, cur_units_x_cons_h,
                               cur_units_x_cons_n * sizeof(int),
                               hipMemcpyHostToDevice, strm_compute_dwt));
  cudaSafeCall(hipMemcpyAsync(unit_vec_vars_d, unit_vec_vars_h,
                               (n_units+1) * N_VEC_VARS * sizeof(float),
                               hipMemcpyHostToDevice, strm_compute_dwt));

  //  Invoke kernel
  Kernel_Compute_dWt_TICtxt<<<cur_units_x_cons_n, n_threads, 0, strm_compute_dwt>>>
    (cur_units_x_cons_d, unit_vec_vars_d, con_params_d, units_d,
     own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, n_units+1);

  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_compute_dwt));
  }
}



__global__ void Kernel_Compute_Weights
(float* own_cons_mem_d, bigint* con_mem_idxs_d, int* con_allocs_d, int* con_sizes_d,
 float* wt_sig_fun_d) {
  const int ucidx = blockIdx.x;  // full unit x con idx here
  const int nth = blockDim.x;

  const int sz = con_sizes_d[ucidx];
  float* wts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::WT));
  float* dwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::DWT));
  float* fwts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::FWT));
  float* swts = own_cons_mem_d + con_mem_idxs_d[ucidx] +
    (con_allocs_d[ucidx] * (1 + LeabraConSpecCuda::SWT));

  int th = threadIdx.x;
  const float cn_per_th = ((float)sz / (float)nth);
  int st = __float2int_rn((float)th * cn_per_th);
  int ed = __float2int_rn((float)(th+1) * cn_per_th);
  //  ed = ed < sz ? ed : sz;     // max of sz
  while(st < ed) {
    float& dwt = dwts[st];
    if(dwt != 0.0f) {
      float& wt = wts[st];
      float& fwt = fwts[st];
      float& swt = swts[st];
      if(dwt > 0.0f)  dwt *= (1.0f - fwt);
      else            dwt *= fwt;
      fwt += dwt;
      swt = fwt;                // keep sync'd -- not tech necc..

      int idx = __float2int_rd(fwt * 10000.0f); // sig_res_inv
      wt = wt_sig_fun_d[idx];

      dwt = 0.0f;
    }
    st++;
  }
}

void LeabraConSpecCuda::Compute_Weights(bool sync) {
  //  Invoke kernel -- does all
  Kernel_Compute_Weights<<<own_units_x_cons, n_threads, 0, strm_compute_wt>>>
    (own_cons_mem_d, con_mem_idxs_d, con_allocs_d, con_sizes_d, wt_sig_fun_d);

  if(sync) {
    cudaSafeCall(hipStreamSynchronize(strm_compute_wt));
  }
}


