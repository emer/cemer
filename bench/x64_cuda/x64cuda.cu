#include "hip/hip_runtime.h"
// Copyright, 1995-2013, Regents of the University of Colorado,
// Carnegie Mellon University, Princeton University.
//
// This file is part of Emergent
//
//   Emergent is free software; you can redistribute it and/or modify
//   it under the terms of the GNU General Public License as published by
//   the Free Software Foundation; either version 2 of the License, or
//   (at your option) any later version.
//
//   Emergent is distributed in the hope that it will be useful,
//   but WITHOUT ANY WARRANTY; without even the implied warranty of
//   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.

#include "x64.h"

texture<float,1> tex_x_float;
inline void bind_x(const float * x)
{
  size_t offset = size_t(-1);
  cudaSafeCall(hipBindTexture(&offset, tex_x_float, x));
}

inline void unbind_x(const float * x)
{
  cudaSafeCall(hipUnbindTexture(tex_x_float));
}

__inline__ __device__ float fetch_x(const int& i, const float * x)
{
  return tex1Dfetch(tex_x_float, i);
}


float rand_float() {
  float r = (float)rand()/(float)RAND_MAX;
  return r;
}


//////////////////////////////////////////////////
//      LeabraNetwork

LeabraNetwork::LeabraNetwork() {
  n_units = 0;
  units_flat = NULL;
  send_netin_tmp = NULL;
}

LeabraNetwork::~LeabraNetwork() {
  FreeUnits();
}

void LeabraNetwork::BuildUnits(int n_un) {
  FreeUnits();
  if(n_un == 0) return;
  n_units = n_un;
  units_flat = (LeabraUnit**)calloc(n_units, sizeof(LeabraUnit*));
  send_netin_tmp = (float*)calloc(n_units, sizeof(float));

  for(int i=0; i<n_units; i++) {
    units_flat[i] = new LeabraUnit;
  }
}

void LeabraNetwork::FreeUnits() {
  for(int i=0; i<n_units; i++) {
    delete units_flat[i];
  }

  if(units_flat) { free(units_flat); units_flat = NULL; }
  if(send_netin_tmp) { free(send_netin_tmp); send_netin_tmp = NULL; }
  n_units = 0;
}

void LeabraNetwork::ConnectUnits(int n_per_un, int n_layers, int n_prjns) {
  int n_per_lay = n_units / n_layers;
  int n_per_prjn = n_per_un / n_prjns;
  while(n_per_lay < n_per_prjn) { // make sure it fits
    n_prjns++;
    n_per_prjn = n_per_un / n_prjns;
    std::cerr << "incremeted prjns to: " << n_prjns << std::endl;
  }

  for(int i=0; i<n_units; i++) {
    LeabraUnit* un = units_flat[i];
    un->send[0].AllocCons(n_per_un);

    int lay_no = i / n_per_lay;
    int lay_pos = i % n_per_lay;

    for(int j=0; j<n_prjns; j++) {
      int to_lay;
      if(j == 0)
        to_lay = (lay_no - 1);
      else
        to_lay = lay_no + j;
      if(to_lay < 0) to_lay += n_layers;
      if(to_lay >= n_layers) to_lay -= n_layers;

      int to_lay_st = to_lay * n_per_un + lay_pos; // start at our offset in layer
      for(int k=0; k<n_per_prjn; k++) {
        int to_idx = (to_lay_st + k) % n_units;
        LeabraCon* cn = un->send[0].ConnectUnOwnCn(to_idx);
        cn->wt = rand_float();
      }
    }
  }
}

void LeabraNetwork::Send_Netin(float pct_delta) {
  static bool did_stats = false;
  // init tmp
  for(int i=0; i<n_units; i++) {
    send_netin_tmp[i] = 0.0f;
  }

  // todo: unsuccessful attempt to focus warnings..
// #pragma clang diagnostic push
// #pragma clang diagnostic warning "-Weverything"
  // send to send_netin_tmp

  // And now for the kernel 
  int n_send = (int)(pct_delta * n_units);
  for(int s = 0; s<n_send; s++) {
    int si = rand() % n_units;
    LeabraUnit* un = units_flat[si];
    float su_act_delta = rand_float(); // fake
    un->send[0].Send_NetinDelta(this, su_act_delta);
  }

// #pragma clang diagnostic pop

  // integrate changes from tmp

  int n_non_zero = 0;
  for(int i=0; i<n_units; i++) {
    // note: actual one has sub-loop per unit
    LeabraUnit* un = units_flat[i];
    float nd = send_netin_tmp[i];
    if(nd != 0.0f)
      n_non_zero++;
    un->net_delta = nd;
    // todo: could do further things here to simulate more computation
  }

  if(!did_stats) {
    std::cout << "n_non_zero: " << n_non_zero << " of total: " << n_units << std::endl;
    did_stats = true;
  }
}


/*
 * In this method, I build all the relevant data needed to run the sender calculation
 * on the GPU. This code proceeds in several steps:
 *
 * (1) First is builds a CSC (Compressed Sparse Column) matrix for doing the calculation
 * doing the update calculation. Imagine the following connectivity pattern:
 *
 * Input Neuron |  1      |   2      |  3    ...
 * ====================================================
 * Output    1  |  I      |   w_2_1  |  0
 *           2  |  w_1_2  |   I      |  0
 *           3  |  0      |   0      |  I
 *           4  |  w_1_4  |   w_2_4  |  w_3_4
 *           5  |  w_1_5  |   w_2_5  |  0
 *           6  |  w_1_6  |   0      |  0
 *           7  |  0      |   0      |  w_3_7
 *           8  |  0      |   0      |  0
 *           9  |  0      |   w_2_9  |  w_3_9
 *          10  |  w_1_10 |   w_2_10 |  w_3_10
 *
 * In the compressed sparse column form, the 0's and I's are not stored. There are three
 * arrays describing the compressed sparse column form of the matrix:
 *  (a) column_offsets
 *  (b) row_indices_offsets
 *  (c) weights
 * Each of these is a linear array in memory. The column_offsets is smaller vector
 * where each entry in gives the location of the start of a new column of data in the
 * row_indices and weights data structures. The first value in column_offsets is 0.
 * The last entry is the total number of entries in the column_indices and/or weights
 * vectors. column_offsets should have length n_units+1.
 *
 * Thus, in the array above, we  have
 *   column_offsets = [0, 5, 10, 14]
 *   row_indices    = [2, 4, 5, 6, 10, 1, 4, 5, 9, 10, 4, 7, 9, 10]
 *   weights        = [w_1_2, w_1_4, w_1_5, w_1_6, w_1_10, w_2_1, w_2_4, w_2_5, w_2_9, w_2_10, w_3_4, w_3_7, w_3_9, w_3_10]
 *
 * (2) Once the matrix is built, we do a calculation of the maximum number of connections 
 * in a column. This will prove useful later on as we add padding to the columns in 
 * order to get aligned memory. Currently, this is not used.
 *
 * (3) Finally, once this array is built on the CPU, we then allocate space on the GPU
 * for this data as well for several auxilliary data structures needed in the calculation.
 * 
 */
void LeabraNetwork::BuildCudaData(float pct_delta) {
  totalSize = 0;
  std::vector<int> column_offsets_cpu(n_units+1);
  std::fill(column_offsets_cpu.begin(), column_offsets_cpu.end(), 0);
  std::vector<int> column_count_cpu(n_units);
  std::fill(column_count_cpu.begin(), column_count_cpu.end(), 0);
  std::vector<int> row_indices_cpu(0);
  std::vector<float> wts_cpu(0);
  for(int s = 0; s<n_units; s++) {
    LeabraUnit* un = units_flat[s];    
    totalSize += un->send->size;
    column_offsets_cpu[s+1]=column_offsets_cpu[s]+un->send->size;
    column_count_cpu[s]=un->send->size;
    for(int i = 0; i<un->send->size; i++) {
      row_indices_cpu.push_back(un->send->UnIdx(i));
      wts_cpu.push_back(un->send->OwnCn(i)->wt);
    }
  }

  /* calculate the maximum number of entries in a column */
  std::vector<int>::iterator itmin = std::min_element(column_count_cpu.begin(),column_count_cpu.end());
  std::vector<int>::iterator itmax = std::max_element(column_count_cpu.begin(),column_count_cpu.end());
  maxConnections = *itmax;
  printf("%d %d min=%d max=%d\n",totalSize,row_indices_cpu.size(),*itmin,*itmax);

  /* Now, send the data to the GPU */
  cudaSafeCall(hipMalloc((void **)&column_offsets_gpu,(n_units+1)*sizeof(int)));
  cudaSafeCall(hipMemcpy(column_offsets_gpu,&(column_offsets_cpu[0]),(n_units+1)*sizeof(int),hipMemcpyHostToDevice));

  cudaSafeCall(hipMalloc((void **)&row_indices_gpu,totalSize*sizeof(int)));
  cudaSafeCall(hipMemcpy(row_indices_gpu,&(row_indices_cpu[0]),totalSize*sizeof(int),hipMemcpyHostToDevice));

  cudaSafeCall(hipMalloc((void **)&wts_gpu,totalSize*sizeof(float)));
  cudaSafeCall(hipMemcpy(wts_gpu,&(wts_cpu[0]),totalSize*sizeof(float),hipMemcpyHostToDevice));

  /* Here I tried to put the weights in a texture. I thought this might give faster memory accesses.
     However since the reads into the wts vector is pretty much coalesced, this doesn't make any 
     real difference. Textures are useful when nearby threads on the GPU reads (not writes) data that 
     are scattered in memory. This may be useful later so we leave it in. */
  //bind_x(wts_gpu);

  /* CPU and GPU data structures for the send_netin vector */
  cudaSafeCall(hipMalloc((void **)&send_netin_gpu,pct_delta*n_units*n_units*sizeof(float)));
  cudaSafeCall(hipHostMalloc((void **)&send_netin_cpu,n_units*sizeof(float)));

  /* Set up CURAND. Currently, this isn't used but if we did want to generate random numbers on the
   * GPU, we would need one of these generators. Should be error-checked. */
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  /* CPU and GPU data structures for the data which is randomly generated. This will not be needed
     for real simulations if a CUDA random number generator is ued. However, for the time being it 
     is useful to have these for testing correctness between CPU and GPU implementations */
  cudaSafeCall(hipMalloc((void **)&su_act_delta_gpu,pct_delta*n_units*sizeof(float)));
  cudaSafeCall(hipHostMalloc((void **)&su_act_delta_cpu,pct_delta*n_units*sizeof(float)));
  cudaSafeCall(hipMalloc((void **)&si_gpu,pct_delta*n_units*sizeof(float)));
  cudaSafeCall(hipHostMalloc((void **)&si_cpu,pct_delta*n_units*sizeof(float)));

  /* Build streams needed for asynchronous execution */
  cudaSafeCall(hipStreamCreate(&stream1));
  cudaSafeCall(hipStreamCreate(&stream2));

  /* Build events needed for timing */
  cudaSafeCall(hipEventCreate(&event1));
  cudaSafeCall(hipEventCreate(&event2));
  totalTime=0.0;
}

/*
 * Here, we free all the GPU data.
 */
void LeabraNetwork::FreeCudaData() {
  cudaSafeCall(hipFree(column_offsets_gpu));
  cudaSafeCall(hipFree(row_indices_gpu));
  //unbind_x(wts_gpu);
  cudaSafeCall(hipFree(wts_gpu));

  cudaSafeCall(hipFree(send_netin_gpu));
  cudaSafeCall(hipHostFree(send_netin_cpu));

  cudaSafeCall(hipFree(su_act_delta_gpu));
  cudaSafeCall(hipHostFree(su_act_delta_cpu));
  cudaSafeCall(hipFree(si_gpu));
  cudaSafeCall(hipHostFree(si_cpu));

  // Free events
  cudaSafeCall(hipEventDestroy(event1));
  cudaSafeCall(hipEventDestroy(event2));

  // Free streams
  cudaSafeCall(hipStreamDestroy(stream1));
  cudaSafeCall(hipStreamDestroy(stream2));
  hiprandDestroyGenerator(gen);
}

#ifndef USE_ATOMICS
#define USE_ATOMICS
#endif
//#undef USE_ATOMICS

/*
 * This kernel initializes an array to some value. Typically, this would be done with 
 * hipMemset. However, as I wanted to be able to run this concurrently with data
 * transfers via hipMemcpyAsync, I decided to write my own kernel for this.
 */
__global__ void kernel0(int n, float val, float *x) {
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if (tid<n) { x[tid]=val; }
}

/*
 * This version of the send kernel updates the values in send_netin with either 
 * (1) atomics or (2) writing to unique data structure and accumulating with
 * a second kernel (which is kernel2). When using (1), kernel2 is not required.
 * Nonetheless, it works and gives a path forward for further improvement.
 */
__global__ void kernel1(const int n_units, const float * su_act_delta, const int * rand_indices, 
			const int * column_offsets, const int * row_indices, const float * wts, float * send_netin1) {

  __shared__ int column;
  __shared__ int cols[2];
  __shared__ float su_act;
  if (threadIdx.x<2) {
    if (threadIdx.x==0) {
      atomicExch(&column, rand_indices[blockIdx.y]); //(int)floor(rand_indices[blockIdx.y]*((float)n_units));
      atomicExch(&su_act,su_act_delta[blockIdx.y]);
    }
    cols[threadIdx.x] = column_offsets[column+threadIdx.x];
  }
  __syncthreads();

  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid<cols[1]-cols[0]) {
    int index = cols[0]+tid;
    int row=row_indices[index];
    //float w=fetch_x(cols[0]+tid,wts);
    float w = wts[index];
#ifdef USE_ATOMICS
    atomicAdd(&(send_netin1[row]),w*su_act);
#else
    send_netin1[blockIdx.y*n_units+row]=w*su_act;
#endif
  }
}

__global__ void kernel2(const int n_units, const int n_send, float * send_netin_gpu) {

  __shared__ volatile float sdata[128];
  int tid = threadIdx.y*blockDim.x+threadIdx.x;
  sdata[tid] = 0.0;
  __syncthreads();

  int tidx = blockIdx.x*blockDim.x + threadIdx.x;
  int tidy = tidx + threadIdx.y*n_units;
  int i;
  if (tidx<n_units) {
    for (i=0; i<n_send; i+=4) {
      sdata[tid] += send_netin_gpu[tidy + i*n_units];
    }
    i-=4;
    if (threadIdx.y<n_send-i) {
      sdata[tid] += send_netin_gpu[tidy + i*n_units];
    }
  }
  __syncthreads();
  if (tidx<n_units) {
    if (threadIdx.y<2) sdata[tid]+=sdata[tid+2];
    if (threadIdx.y<1) send_netin_gpu[tidx] = sdata[tid]+sdata[tid+1];
  }
}

#ifndef COMPARE_RESULTS
#define COMPARE_RESULTS
#endif
#undef COMPARE_RESULTS

void LeabraNetwork::Send_NetinCuda(float pct_delta) {
  static bool did_stats = false;
  // init tmp
  int n_send = (int)(pct_delta * n_units);

  for(int s = 0; s<n_send; s++) {
    si_cpu[s] = rand() % n_units;
    su_act_delta_cpu[s] = 1.0; //rand_float(); // fake
  }

  cudaSafeCall(hipMemcpyAsync(su_act_delta_gpu,su_act_delta_cpu,n_send*sizeof(float),hipMemcpyHostToDevice, stream1));
  cudaSafeCall(hipMemcpyAsync(si_gpu,si_cpu,n_send*sizeof(int),hipMemcpyHostToDevice, stream1));

  int nThreads=128;
  int nBlocks=int(ceil(((float)n_units)/(float)nThreads));
#ifdef USE_ATOMICS
  kernel0<<<nBlocks,nThreads,0,stream2>>>(n_units, 0.0, send_netin_gpu);
#else
  kernel0<<<nBlocks,nThreads,0,stream2>>>(n_send*n_units, 0.0, send_netin_gpu);
#endif
  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipStreamSynchronize(stream1));

  nBlocks=int(ceil(((float)maxConnections)/(float)nThreads));
  dim3 grid(nBlocks,n_send,1);

  cudaSafeCall(hipEventRecord(event1,stream2));
  kernel1<<<grid,nThreads,0,stream2>>>(n_units, su_act_delta_gpu, si_gpu, column_offsets_gpu, row_indices_gpu, wts_gpu, send_netin_gpu);
  cudaSafeCall(hipGetLastError());
  cudaSafeCall(hipEventRecord(event2,stream2));
  cudaSafeCall(hipEventSynchronize(event2));
  float dt=0.0;
  cudaSafeCall(hipEventElapsedTime(&dt,event1,event2));
  totalTime+=dt;

#ifdef USE_ATOMICS

#else
  nThreads=32;
  dim3 block(nThreads,4,1);
  nBlocks=int(ceil(((float)n_units)/(float)nThreads));
  kernel2<<<nBlocks,block,0,stream2>>>(n_units, n_send, send_netin_gpu);
  cudaSafeCall(hipGetLastError());
#endif

  cudaSafeCall(hipMemcpyAsync(send_netin_cpu,send_netin_gpu,n_units*sizeof(float),hipMemcpyDeviceToHost,stream2));
  cudaSafeCall(hipStreamSynchronize(stream2));

#ifdef COMPARE_RESULTS
  // integrate changes from tmp
  for(int i=0; i<n_units; i++) {
    send_netin_tmp[i] = 0.0f;
  }
  for(int s = 0; s<n_send; s++) {
    //int si = rand() % n_units;
    int si = si_cpu[s];
    LeabraUnit* un = units_flat[si];
    //float su_act_delta = rand_float(); // fake
    float su_act_delta = su_act_delta_cpu[s]; // fake
    un->send[0].Send_NetinDelta(this, su_act_delta);
  }  

  for(int i=0; i<n_units; i++) {
    float error = (send_netin_tmp[i]-send_netin_cpu[i])/send_netin_cpu[i];
    if (error>1.e-5)
      printf("%d : cpu=%1.6g\tgpu=%1.6g\terror=%1.6g\n",i,send_netin_tmp[i],send_netin_cpu[i],error);
  }
  exit(0);
#endif

// #pragma clang diagnostic pop
  int n_non_zero = 0;
  for(int i=0; i<n_units; i++) {
    // note: actual one has sub-loop per unit
    LeabraUnit* un = units_flat[i];
    float nd = send_netin_cpu[i];
    if(nd != 0.0f)
      n_non_zero++;
    un->net_delta = nd;
    // todo: could do further things here to simulate more computation
  }

  if(!did_stats) {
    std::cout << "n_non_zero: " << n_non_zero << " of total: " << n_units << std::endl;
    did_stats = true;
  }
}



//////////////////////////////////////////////////
//      LeabraUnit

LeabraUnit::LeabraUnit() {
  send = new LeabraSendCons;    // just make one!
}

LeabraUnit::~LeabraUnit() {
  delete send;
}

//////////////////////////////////////////////////
//      LeabraSendCons

LeabraSendCons::LeabraSendCons() {
  size = 0; alloc_size = 0; other_idx = 0;
  con_size = sizeof(LeabraCon);
  cons_own = NULL;  unit_idxs = NULL;
}
  
LeabraSendCons::~LeabraSendCons() {
  FreeCons();
}

void LeabraSendCons::AllocCons(int sz) {
  if(sz == alloc_size) return;
  FreeCons();
  alloc_size = sz;
  if(alloc_size == 0) return;
  cons_own = (char*)calloc(alloc_size, con_size); // clear it
  unit_idxs = (int*)calloc(alloc_size, sizeof(int));
}

void LeabraSendCons::FreeCons() {
  if(cons_own) { free(cons_own); cons_own = NULL; }
  if(unit_idxs) { free(unit_idxs); unit_idxs = NULL; }
  size = 0;
  alloc_size = 0;
}

LeabraCon* LeabraSendCons::ConnectUnOwnCn(int fm_idx) {
  if(size >= alloc_size) return NULL;
  LeabraCon* rval = OwnCn(size);
  unit_idxs[size++] = fm_idx;
  return rval;
}


//////////////////////////////////////////////////
//      main

int main(int argc, char* argv[]) {

  
  int n_units = atoi(argv[1]);
  int n_per_un = atoi(argv[2]);
  int GPU = atoi(argv[3]);
  int n_layers = 5;
  int n_prjns = 2;

  int n_trials = 100;
  int n_epochs = 5;
  int cyc_per_trl = 70;

  float pct_delta = 0.02f;

  // cuda stuff for testing
  int devices;
  cudaSafeCall(hipGetDeviceCount(&devices));
  printf("There are %d devices\n",devices);

  // todo: get all the args

  int tot_cyc = cyc_per_trl * n_trials * n_epochs;

  int szun = sizeof(LeabraUnit);
  int trgsz = 768;
  if(szun != trgsz) {
    std::cout << "LeabraUnit size is: " << szun << " should be: " << trgsz << std::endl;
  }

  LeabraNetwork net;
  net.BuildUnits(n_units);
  net.ConnectUnits(n_per_un, n_layers, n_prjns);

  net.BuildCudaData(pct_delta);

  TimeUsedHR time_used;
  time_used.StartTimer(true);
  
  for(int cyc = 0; cyc < tot_cyc; cyc++) {
    if (GPU)
      net.Send_NetinCuda(pct_delta);
    else
      net.Send_Netin(pct_delta);
  }

  time_used.EndTimer();

  printf("Average Time Per Send Call(seconds) on the GPU=%g\n",.001*net.totalTime/tot_cyc);
  printf("Total Number of Calls=%d\n",tot_cyc);

  double netin_flops = 2.0;
  double n_flops = (double)pct_delta * (double)n_units * (double)tot_cyc * n_per_un * netin_flops;

  double mflops_sec = (n_flops / time_used.s_used) / 1000000.0;

  std::cout << "total time used: " << time_used.s_used
            << " total flops: " << n_flops
            << " mflops/sec: " << mflops_sec << std::endl;

  net.FreeCudaData();
}

/* template<unsigned int THREADS_PER_ROW> */
/* __global__ void kernel2(int n_units, int n_send, float * rand_indices, float * send_netin_gpu, float * send_netin_small_gpu) { */

/*   extern __shared__ volatile float sdata[]; */
/*   int index = max((int)floor(rand_indices[threadIdx.x]*n_units),0); */
/*   if (threadIdx.x<n_send) sdata[threadIdx.x] = send_netin_gpu[blockIdx.x*n_units + index]; */
/*   __syncthreads(); */
  
/*   // reduce local sums to row sum                                                        */
/*   if (THREADS_PER_ROW > 32) sdata[threadIdx.x] += sdata[threadIdx.x + 32];                                                                                                        */
/*   __syncthreads(); */
/*   if (THREADS_PER_ROW > 16) sdata[threadIdx.x] += sdata[threadIdx.x + 16]; */
/*   if (THREADS_PER_ROW >  8) sdata[threadIdx.x] += sdata[threadIdx.x +  8]; */
/*   if (THREADS_PER_ROW >  4) sdata[threadIdx.x] += sdata[threadIdx.x +  4]; */
/*   if (THREADS_PER_ROW >  2) sdata[threadIdx.x] += sdata[threadIdx.x +  2]; */
/*   if (THREADS_PER_ROW >  1) sdata[threadIdx.x] += sdata[threadIdx.x +  1]; */
  
/*   if (threadIdx.x==0)  */
/*     send_netin_small_gpu[blockIdx.x] = sdata[0]; */
/* } */

/* __global__ void kernel1(int N, int n_units, int n_send, float su_act_delta, float * rand_indices,  */
/* 			       int * column_offsets, int * row_indices, float * wts, float * send_netin) { */

/*   __shared__ int column; */
/*   if (threadIdx.x==0) */
/*     column = max((int)floor(rand_indices[blockIdx.y]*n_units),0); */
  
/*   __syncthreads(); */
/*   int thisColumn = column_offsets[column]; */
/*   int nextColumn = column_offsets[column+1]; */

/*   int tid = blockIdx.x*blockDim.x + threadIdx.x; */
/*   if (tid<nextColumn-thisColumn) { */
/*     int row=row_indices[thisColumn+tid]; */
/*     send_netin[blockIdx.y + row] = wts[row]*su_act_delta; */
/*   } */
/* } */

// generate random numbers
//hipMemset(send_netin_gpu,0,n_units*n_send*sizeof(float));
//hiprandGenerateUniform(gen, rand_indices_gpu, n_send);
// And now for the kernel 
//float su_act_delta = rand_float(); // BS for now
// collect based on the random indices
//kernel1<<<grid,nThreads>>>(totalSize, n_units, n_send, su_act_delta, rand_indices_gpu, column_offsets_gpu, row_indices_gpu, wts_gpu, send_netin_gpu);
//kernel2<64><<<n_units,64,96*sizeof(float)>>>(n_units, n_send, rand_indices_gpu, send_netin_gpu, send_netin_small_gpu);
//err = hipGetLastError();
//if (err!= hipSuccess) { printf("Kernel2 Error = %s at line %d",hipGetErrorString(err),__LINE__); exit(0); }
