#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "embench_cuda.h"

/* Common elements (send and recv)
*/

uint	h_n_units; // this is the requested # units
uint	h_n_units_adj; // adjusted number of units (only used by us)
__constant__
float		c_acts[MAX_UNITS]; // only [h_n_units] init'ed/used
float*		d_nets; // [h_n_units];


extern "C" {

int cuCpHD_Acts(const float* acts) {
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), acts, h_n_units * sizeof(float)));
  return 0;
}

int cuCpDH_Nets(float* nets) {
  CUDA_SAFE_CALL(hipMemcpy((void*)nets, (void*)d_nets,
    h_n_units * sizeof(float), hipMemcpyDeviceToHost));
  return 0;
}

}; // extern "C"


/* Receiver-based

Description of Algorithm

This is a receiver-based algorithm.
Each thread computes partial d_nets inputs for a single Unit. 
Each such partial computation is called a "chunk" -- there
are 32 weights per chunk; therefore, each projection is
represented in atoms of chunks; any unused are set to 0:0.f
(i.e. Unit 0 is also a dummy.)

The Warp size (RCV_N_THREADS) is used to make sure that different
units are processed in parallel-- the allocation algorithm 
insures that each warp uses a different d_nets unit, so the write
updates do not need to be syncronized. 
*/

// Types

typedef int con_si_blk_t[RCV_N_THREADS];
typedef float con_wt_blk_t[RCV_N_THREADS];

// Global data structures

uint	n_cons_pu; 
uint	n_blks; // number of [RCV_N_THREADS] blocks needed

con_si_blk_t*	d_con_si_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]
con_wt_blk_t*	d_con_wt_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]

// can compute act in place, in the net var
__global__ 
void kComputeActs(
  const float* d_nets,
  float* d_acts
) {
  int un_idx = blockIdx.x * blockDim.x + threadIdx.x;
  float tact =  1.0f / (1.0f + expf(-d_acts[un_idx]));
  __syncthreads();
  d_acts[un_idx] = tact; 
}

__global__ 
void kRecv_Netin(
  int n_cons_pu,
  float* d_nets,
  con_si_blk_t* d_con_si_blks,
  con_wt_blk_t* d_con_wt_blks
) {
  // block/thread indexes, for clarity
  int blk = blockIdx.x;
  int tx = threadIdx.x;
  int blki = blk * n_cons_pu;
  int un_idx = blk * blockDim.x + tx;
  float tnet = 0.0f;
  for (int ci = 0; ci < n_cons_pu; ci++, blki++) {
    tnet += c_acts[d_con_si_blks[blki][tx]] * d_con_wt_blks[blki][tx]; 
  }
  d_nets[un_idx] = tnet; 
}

extern "C" {

int cuRecv_AllocMem(uint n_units_, uint n_cons_pu_) {
  if ((n_units_ == 0) || (n_cons_pu_ == 0))
    return 1;
    
  // round up number of units needed to even processable number
  h_n_units_adj = (n_units_ + (RCV_N_THREADS - 1)) & ~(RCV_N_THREADS - 1);
  if ((n_units_ > MAX_UNITS) || (n_cons_pu_ > RCV_MAX_CON_SZ))
    return 2;
    
  h_n_units = n_units_;
  n_cons_pu = n_cons_pu_;
  n_blks = h_n_units / RCV_N_THREADS; // s/b min 1
  
  CUT_DEVICE_INIT();
  
  uint mem_size;
  // init used acts to 0 
  mem_size = n_units_ * sizeof(float);
  void* mem = calloc(n_units_, sizeof(float));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), mem, mem_size));
  free(mem);
  // alloc nets (no init, since overwritten anyway)
  // we allocate dummys if needed
  mem_size = h_n_units * sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_nets, mem_size));
  
  // allocate cons, including for dummys (just have si=0,wt=0)
  mem_size = n_blks * n_cons_pu * sizeof(con_si_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_si_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_si_blks, 0, mem_size));
  mem_size = n_blks * n_cons_pu * sizeof(con_wt_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_wt_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_wt_blks, 0, mem_size));
  
  //TODO: error?
  return 0;
}

int cuRecv_FreeMem() {
  CUDA_SAFE_CALL(hipFree((void*) d_con_wt_blks));
  d_con_wt_blks = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_con_si_blks));
  d_con_si_blks = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_nets));
  d_nets = NULL;

  return 0;
}

int cuRecv_CpHD_Cons(cbGetCon GetCon) {
  int n_con_blks = n_blks * n_cons_pu;
  con_si_blk_t* con_si_buf = (con_si_blk_t*)calloc(n_con_blks, sizeof(con_si_blk_t));
  con_wt_blk_t* con_wt_buf = (con_wt_blk_t*)calloc(n_con_blks, sizeof(con_wt_blk_t));

// we work one silo at a time, each silo is a warp thread
  for (int silo = 0; silo < RCV_N_THREADS; silo++) {
    int blk = 0; // block x
    // note: only iterate actual units, dummys are defaulted to 0,0
    for (int un_idx = silo; un_idx < h_n_units; un_idx += RCV_N_THREADS, blk++) {
      int blki = blk*n_cons_pu;
      for (int con_idx = 0; con_idx < n_cons_pu; con_idx++, blki++) {
        int* con_snd_idx = &(con_si_buf[blki][silo]);
        float* con_wt = &(con_wt_buf[blki][silo]);
        GetCon(un_idx, con_idx, con_snd_idx, con_wt);
      }
    }
  }
  CUDA_SAFE_CALL(hipMemcpy(d_con_si_blks, con_si_buf,
     sizeof(con_si_blk_t) * n_con_blks, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_con_wt_blks, con_wt_buf,
     sizeof(con_wt_blk_t) * n_con_blks, hipMemcpyHostToDevice));
//printf("Copied %d chunks (%d expected)\n", chunks_copied, n_con_chunks);  
    
//  free(con_buf);
  free(con_si_buf);
  free(con_wt_buf);
  return 0;
}

void cuRecv_Netin()
{

    // setup execution parameters
    dim3 grid(n_blks, 1, 1);
    dim3 threads(RCV_N_THREADS, 1, 1);
    
    // execute the kernel
    kRecv_Netin<<< grid, threads >>>(
      n_cons_pu, d_nets, d_con_si_blks, d_con_wt_blks);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}


void cuComputeActs(float* acts)
{
    // setup execution parameters
    dim3 grid(n_blks, 1, 1);
    dim3 threads(RCV_N_THREADS, 1, 1);
    
    // execute the kernel -- uses d_nets as the result (allowed)
    kComputeActs<<< grid, threads >>>(
      d_nets, d_nets);

  CUDA_SAFE_CALL(hipMemcpy((void*)acts, (void*)d_nets,
    h_n_units * sizeof(float), hipMemcpyDeviceToHost));
    
    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}



};
