#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "embench_cuda.h"

/*
Description of Algorithm

This is a receiver-based algorithm.
Each thread computes partial net inputs for a single Unit. 
Each such partial computation is called a "chunk" -- there
are 32 weights per chunk; therefore, each projection is
represented in atoms of chunks; any unused are set to 0:0.f
(i.e. Unit 0 is also a dummy.)

The Warp size (N_THREADS) is used to make sure that different
units are processed in parallel-- the allocation algorithm 
insures that each warp uses a different net unit, so the write
updates do not need to be syncronized. 
*/

// Types

typedef struct con {
  int	snd_idx;
  float	wt;
} con_t;


typedef con_t con_chunk_t[CON_CHUNK_SZ];

// Global data structures

__device__
uint	n_units;
//__device__
//int	n_con_blks;
//__device__
uint	n_blks; // number of blocks needed

__constant__
float		acts[MAX_UNITS];
__device__
float		net[MAX_UNITS];

__device__
int*		blk_map[N_BLOCKS][N_THREADS];
__device__
con_chunk_t*	con_blks[N_BLOCKS][N_THREADS];


__global__ 
void kRecv_Netin() {
  // block/thread indexes, for clarity
  int blk = blockIdx.y;
  int bx = blockIdx.x;
  int tx = threadIdx.x;
//TODO: do i need to copy the chunks to local memory???  
  int un_idx = blk_map[blk][bx][tx];
//TEMP: actually use a real idx
//int un_idx = bx*N_THREADS+tx;
  con_chunk_t& con_chunk = con_blks[blk][bx][tx];
  float tnet = 0.0f;
  for (int ci = 0; ci < CON_CHUNK_SZ; ci++) {
    tnet += acts[con_chunk[ci].snd_idx] * con_chunk[ci].wt; 
  }
  net[un_idx] += tnet; 
//TEMP
//net[un_idx] += __int_as_float(tx);
}

extern "C" {

int cuAllocUnits(uint n_units_, uint n_con_chunks_) {
  if ((n_units_ == 0) || (n_con_chunks_ == 0))
    return 1;
   // round up the number of chunks needed to an even processable number
  n_con_chunks_ = (n_con_chunks_ + ((N_BLOCKS*N_THREADS) - 1)) & ~(N_BLOCKS*N_THREADS);
  if ((n_units_ > MAX_UNITS) || (n_con_chunks_ > MAX_CON_CHUNKS))
    return 2;
    
  n_units = n_units_;
  n_blks = n_con_chunks_ / (N_BLOCKS*N_THREADS); // s/b min 1
//printf("cuAllocUnits: n_blks=%d\n",n_blks); 
  uint mem_size = sizeof(con_blks) * n_blks;
  CUDA_SAFE_CALL(hipMalloc((void**) &con_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(con_blks, 0, mem_size));
  
  mem_size = sizeof(blk_map) * n_blks;
  CUDA_SAFE_CALL(hipMalloc((void**) &blk_map, mem_size));
  CUDA_SAFE_CALL(hipMemset(blk_map, 0, mem_size));
    
  // init acts to 0 
  void* mem = calloc(n_units_, sizeof(float));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("acts"), mem, n_units_ * sizeof(float)));
  free(mem);
  
  //TODO: error?
  return 0;
}

int cuCpHD_Acts(float* acts) {
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("acts"), acts, n_units * sizeof(float)));
  return 0;
}

int cuCpHD_Cons(cbGetCon GetCon) {
  con_chunk_t con_buf; // NOTE: maybe too big to safely alloc on stack...
// we work one silo at a time, each silo is a warp thread
  for (int silo = 0; silo < N_THREADS; silo++) {
    int un_idx = silo;
    int by = 0; // block y
    int bx = 0; // block x
    while (un_idx < n_units) {
      int con_idx = 0;
      bool done = false;
      while (!done) {
        int chunk_idx = 0; // note: only advanced if val read, so ==0 means none read
        while (chunk_idx < CON_CHUNK_SZ) {
          con_t& con = con_buf[chunk_idx];
          done = GetCon(un_idx, con_idx, &(con.snd_idx), &(con.wt));
          if (done) break;
          con_idx++;
          chunk_idx++;
        }
        // chunk_idx is now # of cons read
        if (chunk_idx > 0) { // write chunk and blk map entry
          void* ptr;
          int flat_chunk_idx = (((by * N_BLOCKS) + bx) * N_THREADS) + silo;
          // block map guy -- points to this unit
          ptr = (void*)((char*)blk_map + (flat_chunk_idx * sizeof(int)));
          CUDA_SAFE_CALL(hipMemcpy(&un_idx, ptr,
            sizeof(int), hipMemcpyHostToDevice));
          
          // chunk guy -- only write used portion, so leftover stays 0 in device
          ptr = (void*)((char*)con_blks + (flat_chunk_idx * sizeof(con_chunk_t)));
          CUDA_SAFE_CALL(hipMemcpy(&con_buf, ptr,
            sizeof(con_chunk_t) * chunk_idx, hipMemcpyHostToDevice));
            
          // bump block pointers
          if (++bx >= N_BLOCKS) {
            bx = 0;
            by++;
          }
        }
      }
      un_idx += N_THREADS;
    }
  
  }
    
  return 0;
}

int cuCpDH_Nets(float* nets) {
  CUDA_SAFE_CALL(hipMemcpy((void*)nets, &net,
    n_units * sizeof(float), hipMemcpyDeviceToHost));
  return 0;
}

void cuRecv_Netin()
{
//    CUT_DEVICE_INIT();

    // setup execution parameters
    dim3 grid(N_BLOCKS, n_blks, 1);
    dim3 threads(N_THREADS, 1, 1);
    
    // clear nets
    CUDA_SAFE_CALL(hipMemset(&net, 0, n_units * sizeof(float)));
    
    // execute the kernel
    kRecv_Netin<<< grid, threads >>>();

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}


};
