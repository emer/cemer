#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "embench_cuda.h"

/* Common elements (send and recv)
*/

uint	h_n_units; // this is the requested # units
uint	h_n_units_adj; // adjusted number of units (only used by us)
__constant__
float		c_acts[MAX_UNITS]; // only [h_n_units] init'ed/used
float*		d_nets; // [h_n_units];


extern "C" {

int cuCpHD_Acts(const float* acts) {
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), acts, h_n_units * sizeof(float)));
  return 0;
}

int cuCpDH_Nets(float* nets) {
  CUDA_SAFE_CALL(hipMemcpy((void*)nets, (void*)d_nets,
    h_n_units * sizeof(float), hipMemcpyDeviceToHost));
  return 0;
}

}; // extern "C"


/* Receiver-based

Description of Algorithm

This is a receiver-based algorithm.
Each thread computes the d_net input for a single Unit. 

The Warp size (RCV_N_THREADS) is used to make sure that different
units are processed in parallel-- the allocation algorithm 
insures that each warp uses a different d_nets unit, so the write
updates do not need to be syncronized. 
*/

// Types

typedef int con_si_blk_t[RCV_N_THREADS];
typedef float con_wt_blk_t[RCV_N_THREADS];

// Global data structures

uint	n_cons_pu; 
uint	n_blks; // number of [RCV_N_THREADS] blocks needed

con_si_blk_t*	d_con_si_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]
con_wt_blk_t*	d_con_wt_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]

// can compute act in place, in the net var
__global__ 
void kComputeActs(
  const float* d_nets,
  float* d_acts
) {
  int un_idx = blockIdx.x * blockDim.x + threadIdx.x;
  float tact =  1.0f / (1.0f + expf(-d_acts[un_idx]));
  __syncthreads();
  d_acts[un_idx] = tact; 
}

__global__ 
void kRecv_Netin(
  int n_cons_pu,
  float* d_nets,
  con_si_blk_t* d_con_si_blks,
  con_wt_blk_t* d_con_wt_blks
) {
  // block/thread indexes, for clarity
  int blk = blockIdx.x;
  int tx = threadIdx.x;
  int blki = blk * n_cons_pu;
  int un_idx = blk * blockDim.x + tx;
  float tnet = 0.0f;
  for (int ci = 0; ci < n_cons_pu; ci++, blki++) {
    tnet += c_acts[d_con_si_blks[blki][tx]] * d_con_wt_blks[blki][tx]; 
  }
  d_nets[un_idx] = tnet; 
}

extern "C" {

int cuRecv_AllocMem(uint n_units_, uint n_cons_pu_) {
  if ((n_units_ == 0) || (n_cons_pu_ == 0))
    return 1;
    
  // round up number of units needed to even processable number
  h_n_units_adj = (n_units_ + (RCV_N_THREADS - 1)) & ~(RCV_N_THREADS - 1);
  if ((n_units_ > MAX_UNITS) || (n_cons_pu_ > RCV_MAX_CON_SZ))
    return 2;
    
  h_n_units = n_units_;
  n_cons_pu = n_cons_pu_;
  n_blks = h_n_units / RCV_N_THREADS; // s/b min 1
  
  CUT_DEVICE_INIT();
  
  uint mem_size;
  // init used acts to 0 
  mem_size = n_units_ * sizeof(float);
  void* mem = calloc(n_units_, sizeof(float));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), mem, mem_size));
  free(mem);
  // alloc nets (no init, since overwritten anyway)
  // we allocate dummys if needed
  mem_size = h_n_units * sizeof(float);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_nets, mem_size));
  
  // allocate cons, including for dummys (just have si=0,wt=0)
  mem_size = n_blks * n_cons_pu * sizeof(con_si_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_si_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_si_blks, 0, mem_size));
  mem_size = n_blks * n_cons_pu * sizeof(con_wt_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_wt_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_wt_blks, 0, mem_size));
  
  //TODO: error?
  return 0;
}

int cuRecv_FreeMem() {
  CUDA_SAFE_CALL(hipFree((void*) d_con_wt_blks));
  d_con_wt_blks = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_con_si_blks));
  d_con_si_blks = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_nets));
  d_nets = NULL;

  return 0;
}

int cuRecv_CpHD_Cons(cbGetCon GetCon) {
  int n_con_blks = n_blks * n_cons_pu;
  con_si_blk_t* con_si_buf = (con_si_blk_t*)calloc(n_con_blks, sizeof(con_si_blk_t));
  con_wt_blk_t* con_wt_buf = (con_wt_blk_t*)calloc(n_con_blks, sizeof(con_wt_blk_t));

// we work one silo at a time, each silo is a warp thread
  for (int silo = 0; silo < RCV_N_THREADS; silo++) {
    int blk = 0; // block x
    // note: only iterate actual units, dummys are defaulted to 0,0
    for (int un_idx = silo; un_idx < h_n_units; un_idx += RCV_N_THREADS, blk++) {
      int blki = blk*n_cons_pu;
      for (int con_idx = 0; con_idx < n_cons_pu; con_idx++, blki++) {
        int* con_snd_idx = &(con_si_buf[blki][silo]);
        float* con_wt = &(con_wt_buf[blki][silo]);
        GetCon(un_idx, con_idx, con_snd_idx, con_wt);
      }
    }
  }
  CUDA_SAFE_CALL(hipMemcpy(d_con_si_blks, con_si_buf,
     sizeof(con_si_blk_t) * n_con_blks, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_con_wt_blks, con_wt_buf,
     sizeof(con_wt_blk_t) * n_con_blks, hipMemcpyHostToDevice));
//printf("Copied %d chunks (%d expected)\n", chunks_copied, n_con_chunks);  
    
//  free(con_buf);
  free(con_si_buf);
  free(con_wt_buf);
  return 0;
}

void cuRecv_Netin()
{

    // setup execution parameters
    dim3 grid(n_blks, 1, 1);
    dim3 threads(RCV_N_THREADS, 1, 1);
    
    // execute the kernel
    kRecv_Netin<<< grid, threads >>>(
      n_cons_pu, d_nets, d_con_si_blks, d_con_wt_blks);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}


void cuComputeActs(float* acts)
{
    // setup execution parameters
    dim3 grid(n_blks, 1, 1);
    dim3 threads(RCV_N_THREADS, 1, 1);
    
    // execute the kernel -- uses d_nets as the result (allowed)
    kComputeActs<<< grid, threads >>>(
      d_nets, d_nets);

  CUDA_SAFE_CALL(hipMemcpy((void*)acts, (void*)d_nets,
    h_n_units * sizeof(float), hipMemcpyDeviceToHost));
    
    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}



};


/* Sender-based

Description of Algorithm

This is a sender-based algorithm -- it is significantly more
complex than the receiver-algorithm.

Computation is organized around layers, and then smaller
groups of units from within a layer called a unitgroup (this
is not the same as a UnitGroup in Emergent, but the concept
is similar.)

The sending connections of a unit are organized into a list,
by target unitgroup. Each fixed-size item of this list is
a conchunk.



Each block is dedicated to calculating nets for a set of
unitgroups. Shared memory is used to hold the net values for
that set.

Each block processes a single sending unit's
connections at one time -- since the unit will have a disjoint
set of targets, this insures that each thread will be writing
to a different shared memory net value, so there can be
no conflicts or need for synchronization.
 
The block will loop until all the units that send values to
any of its nets have been finished.

Note that there is not a full mapping from sending unit to 
unitgroup -- for full connections, there will be, but for
partial connection patterns, some ugs will only have some
sending conns from a unit -- in this case there will either
need be dummy targets -- by convention, the dummy neurons
are the topmost ones (at end of act array)

by:
bx:

tx: con_idx

Data:

Runtime layout:

nets [blks][SND_UG_SZ]
*/
/* TODO
// CONSTANTS

#define SND_UG_SZ 

#define NETS_PER_BLOCK 
// Types


// Global data structures

uint	n_cons_pu; 
uint	n_blks; // number of [RCV_N_THREADS] blocks needed

con_si_blk_t*	d_con_si_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]
con_wt_blk_t*	d_con_wt_blks; // [n_blks][n_cons_pu][RCV_N_THREADS]

__global__ 
void kSend_Netin(
  int n_cons_pu,
  float* d_nets,
  con_si_blk_t* d_con_si_blks,
  con_wt_blk_t* d_con_wt_blks
) {
  extern __shared__ float s_nets[]; //[SND_UG_SZ]
  
  // block/thread indexes, for clarity
  int blk = blockIdx.x;
  int tx = threadIdx.x;
  
  // init net
  s_nets[tx] = 0;
  
  int blki = blk * n_cons_pu;
  int un_idx = blk * blockDim.x + tx;
  float tnet = 0.0f;
  for (int ci = 0; ci < n_cons_pu; ci++, blki++) {
    tnet += c_acts[d_con_si_blks[blki][tx]] * d_con_wt_blks[blki][tx]; 
  }
  d_nets[un_idx] = tnet; 
}

*/
