#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "embench_cuda.h"

/*
Description of Algorithm

This is a receiver-based algorithm.
Each thread computes partial net inputs for a single Unit. 
Each such partial computation is called a "chunk" -- there
are 32 weights per chunk; therefore, each projection is
represented in atoms of chunks; any unused are set to 0.

The Warp size is used to make sure that different units
are processed -- the allocation algorithm insures that 
each warp used a different net unit, so the write updates
do not need to be syncronized. 
*/

// Types

typedef struct con {
  int	snd_idx;
  float	wt;
} con_t;


typedef con_t con_chunk_t[CON_CHUNK_SZ];

// Global data structures

__device__
uint	n_units;
//__device__
//int	n_con_blks;
//__device__
uint	n_blks; // number of blocks needed

__constant__
float		acts[MAX_UNITS];
__device__
float		net[MAX_UNITS];

__device__
int*		blk_map[N_BLOCKS][N_THREADS];
__device__
con_chunk_t*	con_blks[N_BLOCKS][N_THREADS];


__global__ 
void kRecv_Netin() {
  // block/thread indexes, for clarity
  int blk = blockIdx.y;
  int bx = blockIdx.x;
  int tx = threadIdx.x;
//TODO: do i need to copy the chunks to local memory???  
//  int un_idx = blk_map[blk][bx][tx];
//TEMP: actually use a real idx
int un_idx = bx*N_THREADS+tx;
  con_chunk_t& con_chunk = con_blks[blk][bx][tx];
  float tnet = 0.0f;
  for (int ci = 0; ci < CON_CHUNK_SZ; ci++) {
    tnet += acts[con_chunk[ci].snd_idx] * con_chunk[ci].wt; 
  }
//  net[un_idx] += tnet; 
//TEMP
net[un_idx] += __int_as_float(tx);
}

extern "C" {

int cuAllocUnits(uint n_units_, uint n_con_chunks_) {
  if ((n_units_ == 0) || (n_con_chunks_ == 0))
    return 1;
   // round up the number of chunks needed to an even processable number
  n_con_chunks_ = (n_con_chunks_ + ((N_BLOCKS*N_THREADS) - 1)) & ~(N_BLOCKS*N_THREADS);
  if ((n_units_ > MAX_UNITS) || (n_con_chunks_ > MAX_CON_CHUNKS))
    return 2;
    
  n_units = n_units_;
  n_blks = n_con_chunks_ / (N_BLOCKS*N_THREADS); // s/b min 1
//printf("cuAllocUnits: n_blks=%d\n",n_blks); 
  uint mem_size = sizeof(con_blks) * n_blks;
  CUDA_SAFE_CALL(hipMalloc((void**) &con_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(con_blks, 0, mem_size));
  
  mem_size = sizeof(blk_map) * n_blks;
  CUDA_SAFE_CALL(hipMalloc((void**) &blk_map, mem_size));
  CUDA_SAFE_CALL(hipMemset(blk_map, 0, mem_size));
    
  // init acts to 0 
  void* mem = calloc(n_units_, sizeof(float));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("acts"), mem, n_units_ * sizeof(float)));
  free(mem);
  
  //TODO: error?
  return 0;
}

int cuCpHD_Acts(float* acts) {
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("acts"), acts, n_units * sizeof(float)));
  return 0;
}

int cuCpHD_Cons(uint un_idx, uint n_cons, cbGetCon GetCon) {
    return 0;
}

int cuCpDH_Nets(float* nets) {
  CUDA_SAFE_CALL(hipMemcpy((void*)nets, &net,
    n_units * sizeof(float), hipMemcpyDeviceToHost));
  return 0;
}

void cuRecv_Netin()
{
//    CUT_DEVICE_INIT();

    // setup execution parameters
    dim3 grid(N_BLOCKS, n_blks, 1);
    dim3 threads(N_THREADS, 1, 1);
    
    // clear nets
    CUDA_SAFE_CALL(hipMemset(&net, 0, n_units * sizeof(float)));
    
    // execute the kernel
    kRecv_Netin<<< grid, threads >>>();

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}


};
