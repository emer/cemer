#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "embench_cuda.h"

/* Common elements (send and recv)
*/

uint	h_n_units;
__constant__
float		c_acts[MAX_UNITS]; // only [h_n_units] init'ed/used
float*		d_nets; // [h_n_units];


extern "C" {

int cuCpHD_Acts(const float* acts) {
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), acts, h_n_units * sizeof(float)));
  return 0;
}

int cuCpDH_Nets(float* nets) {
  CUDA_SAFE_CALL(hipMemcpy((void*)nets, (void*)d_nets,
    h_n_units * sizeof(float), hipMemcpyDeviceToHost));
  return 0;
}

}; // extern "C"


/* Receiver-based

Description of Algorithm

This is a receiver-based algorithm.
Each thread computes partial d_nets inputs for a single Unit. 
Each such partial computation is called a "chunk" -- there
are 32 weights per chunk; therefore, each projection is
represented in atoms of chunks; any unused are set to 0:0.f
(i.e. Unit 0 is also a dummy.)

The Warp size (RCV_N_THREADS) is used to make sure that different
units are processed in parallel-- the allocation algorithm 
insures that each warp uses a different d_nets unit, so the write
updates do not need to be syncronized. 
*/

// Types

typedef struct con {
  int	snd_idx;
  float	wt;
} con_t;


typedef int blk_map_t[RCV_N_THREADS];
//typedef con_t con_blk_t[RCV_CON_CHUNK_SZ][RCV_N_THREADS];
typedef int con_si_blk_t[RCV_CON_CHUNK_SZ][RCV_N_THREADS];
typedef float con_wt_blk_t[RCV_CON_CHUNK_SZ][RCV_N_THREADS];

// Global data structures

//__device__
uint	n_con_chunks; // just for reference
//__device__
uint	n_blks; // number of [RCV_N_THREADS] blocks needed

blk_map_t*	d_blk_map; // [n_blks][RCV_N_THREADS]
//con_blk_t*	d_con_blks; // [n_blks][RCV_N_THREADS]
con_si_blk_t*	d_con_si_blks; // [n_blks][RCV_N_THREADS]
con_wt_blk_t*	d_con_wt_blks; // [n_blks][RCV_N_THREADS]

__global__ 
void kRecv_Netin(
  float* d_nets,
  blk_map_t* d_blk_map,
  con_si_blk_t* d_con_si_blks,
  con_wt_blk_t* d_con_wt_blks
) {
  // block/thread indexes, for clarity
  int blk = blockIdx.x;
  int tx = threadIdx.x;
  int un_idx = d_blk_map[blk][tx];
  float tnet = 0.0f;
  for (int ci = 0; ci < RCV_CON_CHUNK_SZ; ci++) {
/*    con_t& con = d_con_blks[blk][ci][tx];
    tnet += c_acts[con.snd_idx] * con.wt; */
    tnet += c_acts[d_con_si_blks[blk][ci][tx]] * d_con_wt_blks[blk][ci][tx]; 
  }
  d_nets[un_idx] += tnet; 
  __syncthreads();
}

extern "C" {

int cuRecv_AllocMem(uint n_units_, uint silo_sz_) {
  if ((n_units_ == 0) || (silo_sz_ == 0))
    return 1;
    
  // round up number of blks needed to even processable number
  
   // round up the number of chunks needed to an even processable number
  n_con_chunks = (silo_sz_ * RCV_N_THREADS); // for reference
//printf("cuAllocMem: n_con_chunks=%d\n",n_con_chunks); 
  if ((n_units_ > MAX_UNITS) || (silo_sz_ > RCV_MAX_SILO_SZ))
    return 2;
    
  h_n_units = n_units_;
  n_blks = silo_sz_; // s/b min 1
  
  CUT_DEVICE_INIT();
  
  uint mem_size;
  // init used acts to 0 
  mem_size = n_units_ * sizeof(float);
  void* mem = calloc(n_units_, sizeof(float));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("c_acts"), mem, mem_size));
  free(mem);
  // alloc nets (no init, since overwritten anyway)
  CUDA_SAFE_CALL(hipMalloc((void**) &d_nets, mem_size));
  
  
//printf("cuAllocMem: n_blks=%d\n",n_blks); 
  // block map
  mem_size = n_blks * sizeof(blk_map_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_blk_map, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_blk_map, 0, mem_size));
    
/*  mem_size = n_blks * sizeof(con_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_blks, 0, mem_size));*/
  mem_size = n_blks * sizeof(con_si_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_si_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_si_blks, 0, mem_size));
  mem_size = n_blks * sizeof(con_wt_blk_t);
  CUDA_SAFE_CALL(hipMalloc((void**) &d_con_wt_blks, mem_size));
  CUDA_SAFE_CALL(hipMemset(d_con_wt_blks, 0, mem_size));
  
  //TODO: error?
  return 0;
}

int cuRecv_FreeMem() {
  CUDA_SAFE_CALL(hipFree((void*) d_nets));
  d_nets = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_blk_map));
  d_blk_map = NULL;
/*  CUDA_SAFE_CALL(hipFree((void*) d_con_blks));
  d_con_blks = NULL;*/
  CUDA_SAFE_CALL(hipFree((void*) d_con_si_blks));
  d_con_si_blks = NULL;
  CUDA_SAFE_CALL(hipFree((void*) d_con_wt_blks));
  d_con_wt_blks = NULL;

  return 0;
}

int cuRecv_CpHD_Cons(cbGetCon GetCon) {
  int chunks_copied = 0; // sanity check
//  con_blk_t* con_buf = (con_blk_t*)calloc(n_blks, sizeof(con_blk_t));
  con_si_blk_t* con_si_buf = (con_si_blk_t*)calloc(n_blks, sizeof(con_si_blk_t));
  con_wt_blk_t* con_wt_buf = (con_wt_blk_t*)calloc(n_blks, sizeof(con_wt_blk_t));
// we work one silo at a time, each silo is a warp thread
  for (int silo = 0; silo < RCV_N_THREADS; silo++) {
    int un_idx = silo;
    int blk = 0; // block x
    while (un_idx < h_n_units) {
      int con_idx = 0;
      bool done = false;
      while (!done) {
        int chunk_idx = 0; // note: only advanced if val read, so ==0 means none read
        while (chunk_idx < RCV_CON_CHUNK_SZ) {
//          con_t& con = con_buf[blk][chunk_idx][silo];
          int* con_snd_idx = &(con_si_buf[blk][chunk_idx][silo]);
          float* con_wt = &(con_wt_buf[blk][chunk_idx][silo]);
          done = GetCon(un_idx, con_idx, con_snd_idx, con_wt);
          if (done) break;
          con_idx++;
          chunk_idx++;
        }
        // chunk_idx is now # of cons read
        if (chunk_idx > 0) { // write chunk and blk map entry
          void* ptr;
          int flat_chunk_idx = (blk * RCV_N_THREADS) + silo;
          // block map guy -- points to this unit
          ptr = (void*)((char*)d_blk_map + (flat_chunk_idx * sizeof(int)));
          CUDA_SAFE_CALL(hipMemcpy(ptr, &un_idx,
            sizeof(int), hipMemcpyHostToDevice));

          // bump block pointers/counters
          chunks_copied++;
          blk++;
        }
      }
      un_idx += RCV_N_THREADS;
    }
  }
/*  CUDA_SAFE_CALL(hipMemcpy(d_con_blks, con_buf,
     sizeof(con_blk_t) * n_blks, hipMemcpyHostToDevice));*/
  CUDA_SAFE_CALL(hipMemcpy(d_con_si_blks, con_si_buf,
     sizeof(con_si_blk_t) * n_blks, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_con_wt_blks, con_wt_buf,
     sizeof(con_wt_blk_t) * n_blks, hipMemcpyHostToDevice));
//printf("Copied %d chunks (%d expected)\n", chunks_copied, n_con_chunks);  
    
//  free(con_buf);
  free(con_si_buf);
  free(con_wt_buf);
  return 0;
}

void cuRecv_Netin()
{

    // setup execution parameters
    dim3 grid(n_blks, 1, 1);
    dim3 threads(RCV_N_THREADS, 1, 1);
    
    // clear nets
    CUDA_SAFE_CALL(hipMemset(d_nets, 0, h_n_units * sizeof(float)));
    
    // execute the kernel
//    kRecv_Netin<<< grid, threads >>>(d_nets, d_blk_map, d_con_blks);
    kRecv_Netin<<< grid, threads >>>(
      d_nets, d_blk_map, d_con_si_blks, d_con_wt_blks);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");
}


};
